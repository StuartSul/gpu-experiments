#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcf;
template<int D, int NUM_WORKERS> struct attn_fwd_layout {
    using qo_tile   = st_bf<64, D>;
    using kv_tile   = st_bf<D==64?192:128, D>;
    using qo_global = kittens::gl<bf16, -1, -1, -1, D, qo_tile>;
    using kv_global = kittens::gl<bf16, -1, -1, -1, D, kv_tile>;
    struct globals { qo_global O, Q; kv_global K, V; };
    struct input_block    { kv_tile k, v; };
    struct scratch_block  { qo_tile q[NUM_WORKERS]; };
    struct common_state   { int batch, head, seq; };
    struct consumer_state {
        rt_fl<16, qo_tile::cols> o_reg;
        col_vec<rt_fl<16, kv_tile::rows>> max_vec, norm_vec;
        col_vec<rt_fl<16, kv_tile::rows>> max_vec_last_scaled, max_vec_scaled;
        rt_fl<16, kv_tile::rows> att_block;
        rt_bf<16, kv_tile::rows> att_block_mma;
    };
};
template<int D> struct attn_fwd_template {
    static constexpr int NUM_CONSUMER_WARPS = 12, NUM_WORKERS = NUM_CONSUMER_WARPS/4, INPUT_PIPE_STAGES = 2;
    using layout = attn_fwd_layout<D, NUM_WORKERS>;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        int task_id = gridDim.x*args.task_iter + blockIdx.x;
        int seq_q = (args.globals.Q.rows + NUM_WORKERS*layout::qo_tile::rows - 1)/(NUM_WORKERS*layout::qo_tile::rows);
        args.common.batch = task_id / (seq_q*args.globals.K.depth); task_id -= args.common.batch * seq_q * args.globals.K.depth;
        args.common.head  = task_id / seq_q;                        task_id -= args.common.head  * seq_q;
        args.common.seq   = task_id;
        args.num_iters = args.common.batch < args.globals.Q.batch ? (args.globals.K.rows + layout::kv_tile::rows - 1)/(layout::kv_tile::rows) : -1;
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {
            warpgroup::producer_registers();
        }
        __device__ static inline void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                tma::load_async(args.input.k, args.globals.K, {args.common.batch, args.common.head, args.iter, 0}, args.inputs_arrived);
                tma::load_async(args.input.v, args.globals.V, {args.common.batch, args.common.head, args.iter, 0}, args.inputs_arrived);
            }
            else if(laneid() == 0) arrive(args.inputs_arrived);
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {
            warpgroup::consumer_registers<NUM_WORKERS>();
            if((args.common.seq*NUM_WORKERS + warpgroup::groupid())*layout::qo_tile::rows < args.globals.Q.rows) // out of bounds?
                warpgroup::load(args.scratch.q[warpgroup::groupid()], args.globals.Q,
                                {args.common.batch, args.common.head, args.common.seq*NUM_WORKERS+warpgroup::groupid(), 0});
            zero(args.state.o_reg);
            zero(args.state.norm_vec);
            neg_infty(args.state.max_vec);
            warpgroup::sync(warpgroup::groupid());
        }
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            constexpr float TEMPERATURE_SCALE = (D == 128) ? 0.08838834764f*1.44269504089f : 0.125f*1.44269504089f;
            // A = Q @ K.T
            warpgroup::mm_ABt(args.state.att_block, args.scratch.q[warpgroup::groupid()], args.input.k);
            mul(args.state.max_vec_last_scaled, args.state.max_vec, TEMPERATURE_SCALE);
            warpgroup::mma_async_wait();
            // softmax
            right_fill(args.state.att_block, args.state.att_block, args.globals.K.rows - args.iter*layout::kv_tile::rows, base_types::constants<float>::neg_infty());
            row_max(args.state.max_vec, args.state.att_block, args.state.max_vec); // accumulate onto the max_vec
            mul(args.state.max_vec_scaled, args.state.max_vec, TEMPERATURE_SCALE);
            mul(args.state.att_block, args.state.att_block, TEMPERATURE_SCALE);
            sub_row(args.state.att_block, args.state.att_block, args.state.max_vec_scaled);
            exp2(args.state.att_block, args.state.att_block);
            sub(args.state.max_vec_last_scaled, args.state.max_vec_last_scaled, args.state.max_vec_scaled);
            exp2(args.state.max_vec_last_scaled, args.state.max_vec_last_scaled);
            mul(args.state.norm_vec, args.state.norm_vec, args.state.max_vec_last_scaled);
            row_sum(args.state.norm_vec, args.state.att_block, args.state.norm_vec); // accumulate onto the norm_vec
            mul_row(args.state.o_reg, args.state.o_reg, args.state.max_vec_last_scaled); // normalize o_reg before mma
            copy(args.state.att_block_mma, args.state.att_block); // convert to bf16 for mma
            // O += A @ V
            warpgroup::mma_AB(args.state.o_reg, args.state.att_block_mma, args.input.v);
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.inputs_finished); // done!
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            if((args.common.seq*NUM_WORKERS+warpgroup::groupid())*64 < args.globals.Q.rows) { // out of bounds?
                div_row(args.state.o_reg, args.state.o_reg, args.state.norm_vec);
                auto &o_smem = reinterpret_cast<typename layout::qo_tile&>(args.scratch.q[warpgroup::groupid()]);
                warpgroup::store(o_smem, args.state.o_reg);
                warpgroup::sync(warpgroup::groupid());
                if(warpgroup::warpid() == 0)
                    tma::store_async(args.globals.O, o_smem, {args.common.batch, args.common.head, args.common.seq*NUM_WORKERS+warpgroup::groupid(), 0});
                tma::store_async_read_wait();
            }
            __syncwarp();
            if(laneid() == 0) arrive(args.finish_finished); // done!
        }
    };
};
// kernel is kittens::prototype::lcf::kernel<attn_fwd_template<HEAD_DIM>>;


#include <iostream>
#include <string>
#include <fstream>

constexpr int ATTN_B = 256;
constexpr int ATTN_H = 1;
constexpr int ATTN_N = 924; // 768*2; // 4096;
constexpr int ATTN_D = 128; // hardcoded into this kernel
constexpr int ITER   = 10;

#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError( const char *file, const int line ) {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
}

// Compute FLOPs for forward attention
constexpr uint64_t ATTN_FLOPS = 
    2llu * ATTN_B * ATTN_H * ATTN_N * ATTN_N * ATTN_D + // Q * K^T: 2BHNND (multiply-add)
    4llu * ATTN_B * ATTN_H * ATTN_N * ATTN_N +          // Softmax: 2BHNN (exp and divide, plus flash-attn bookkeeping)
    2llu * ATTN_B * ATTN_H * ATTN_N * ATTN_N * ATTN_D;      // (Q * K^T) * V: 2BHNND (multiply-add)

int main(int argc, char **argv) {
    // TODO: consider doing sequential kernel launches to force batches dimension element to execute sequentially,
    // which may increase the probability of L2 cache hits on KV
    using ker_template = attn_fwd_template<ATTN_D>;

    std::cout << "Entered main!" << std::endl;

    // create dummy variables that are the right size
    constexpr int TOTAL_ELEMENTS = ATTN_B*ATTN_H*ATTN_N*ATTN_D;
    constexpr int TOTAL_UNIQUE_ELEMENTS = ATTN_N*ATTN_D*ATTN_H;

    float *q = new float[TOTAL_ELEMENTS];
    float *k = new float[TOTAL_ELEMENTS];
    float *v = new float[TOTAL_ELEMENTS];
    float *o_ref = new float[TOTAL_ELEMENTS];

    bf16 *q_bf = new bf16[TOTAL_ELEMENTS];
    bf16 *k_bf = new bf16[TOTAL_ELEMENTS];
    bf16 *v_bf = new bf16[TOTAL_ELEMENTS];
    bf16 *o_bf = new bf16[TOTAL_ELEMENTS];
    float *o = new float[TOTAL_ELEMENTS];

    std::ifstream infile(argv[1]);

    std::cout << "Starting to enter!" << std::endl;

    for(int i = 0; i < TOTAL_ELEMENTS/ATTN_B; i++) infile >> q[i];
    std::cout << "Finished loading Q" << std::endl;
    for(int i = 0; i < TOTAL_ELEMENTS/ATTN_B; i++) infile >> k[i];
    std::cout << "Finished loading K" << std::endl;
    for(int i = 0; i < TOTAL_ELEMENTS/ATTN_B; i++) infile >> v[i];
    std::cout << "Finished loading V" << std::endl;
    for(int i = 0; i < TOTAL_ELEMENTS/ATTN_B; i++) infile >> o_ref[i];
    std::cout << "Finished loading O_REF" << std::endl;

    std::cout << "Finished loading file from " << argv[1] << "!" << std::endl;

    // replicate into batch elements
    for(int i = 0; i < TOTAL_ELEMENTS; i++) {
        q_bf[i] = __float2bfloat16(q[i % (TOTAL_ELEMENTS/ATTN_B)]);
        k_bf[i] = __float2bfloat16(k[i % (TOTAL_ELEMENTS/ATTN_B)]);
        v_bf[i] = __float2bfloat16(v[i % (TOTAL_ELEMENTS/ATTN_B)]);
    }

    bf16 *d_q, *d_k, *d_v, *d_o;
    hipMalloc(&d_q, TOTAL_ELEMENTS * sizeof(bf16));
    hipMalloc(&d_k, TOTAL_ELEMENTS * sizeof(bf16));
    hipMalloc(&d_v, TOTAL_ELEMENTS * sizeof(bf16));
    hipMalloc(&d_o, TOTAL_ELEMENTS * sizeof(bf16));

    hipMemcpy(d_q, q_bf, TOTAL_ELEMENTS * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_k, k_bf, TOTAL_ELEMENTS * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v_bf, TOTAL_ELEMENTS * sizeof(bf16), hipMemcpyHostToDevice);

    ker_template::layout::qo_global Qg(d_q, ATTN_B, ATTN_H, ATTN_N, nullptr);
    ker_template::layout::kv_global Kg(d_k, ATTN_B, ATTN_H, ATTN_N, nullptr);
    ker_template::layout::kv_global Vg(d_v, ATTN_B, ATTN_H, ATTN_N, nullptr);
    ker_template::layout::qo_global Og(d_o, ATTN_B, ATTN_H, ATTN_N, nullptr);
    ker_template::layout::globals globals = {Og, Qg, Kg, Vg};
    
    unsigned long mem_size = kittens::MAX_SHARED_MEMORY - 2000; // have the flag tell us
    
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        prototype::lcf::kernel<ker_template>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );

    hipDeviceSynchronize();
    std::cout << "Starting kernel\n";
    constexpr int NUM_WORKERS = prototype::detail::NUM_CONSUMER_WARPGROUPS_v<ker_template>;
    constexpr int BLOCK_SIZE = prototype::detail::NUM_THREADS_v<ker_template>;
    dim3 grid(132, 1, 1);
    // dim3 bad_grid(grid.z, grid.y, grid.x);
    std::cout << "Grid size: " << grid.x << " x " << grid.y << " x " << grid.z << std::endl;
    // warmup
    for(int j = 0; j < ITER; j++)
        prototype::lcf::kernel<ker_template><<<grid, BLOCK_SIZE, mem_size>>>(globals);
    hipDeviceSynchronize();
    
    const auto start = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < ITER; i++) {
        prototype::lcf::kernel<ker_template><<<grid, BLOCK_SIZE, mem_size>>>(globals);
    }
    hipDeviceSynchronize();
    const auto finish = std::chrono::high_resolution_clock::now();
    CudaCheckError();
    std::cout << "Finished kernel\n";
    
    // check correctness
    hipMemcpy(o_bf, d_o, TOTAL_ELEMENTS * sizeof(bf16), hipMemcpyDeviceToHost);
    for(int i = 0; i < TOTAL_ELEMENTS; i++) {
        o[i] = __bfloat162float(o_bf[i]);
    }

    bool good = true;
    std::ofstream o_ref_file("printouts/o_ref.txt");
    std::ofstream o_file("printouts/o.txt");
    std::ofstream diff_file("printouts/diff.txt");

    float total_diff = 0;
    float max_error = 0; 

    for(int i = 0; i < TOTAL_ELEMENTS; i++) {
        float diff = o[i] - o_ref[i % (TOTAL_ELEMENTS/ATTN_B)];

        if (i < TOTAL_UNIQUE_ELEMENTS) {
            o_ref_file << o_ref[i % (TOTAL_ELEMENTS/ATTN_B)] << ' ';
            o_file << o[i] << ' ';
            diff_file << diff << ' ';
        }
        if (i % ATTN_D == ATTN_D-1) {
            o_ref_file << '\n';
            o_file << '\n';
            diff_file << '\n';
        }
        if(abs(diff) > 0.01 || isnan(diff)) {
            good = false;
        }

        total_diff += abs(diff);
        if (abs(diff) > max_error) {
            max_error = abs(diff);
        }
    }

    // print average difference
    std::cout << "Average o difference: " << total_diff / TOTAL_ELEMENTS << std::endl;
    std::cout << "Max     o difference: " << max_error << std::endl;
    if (abs(total_diff / TOTAL_ELEMENTS) < 1e-3) {
        good = true;
    }

    std::cout << "Average fwd execution time: " << std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count() / ITER << " us" << std::endl;
    if(good) std::cout << "FWD Correct :)\n";
    else std::cout << "FWD Incorrect :(\n";
    // Compute and print average TFLOPs achieved
    double avg_time_s = (double)(std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count()) / (ITER * 1e6);
    double avg_tflops = (ATTN_FLOPS / avg_time_s) / 1e12;
    std::cout << "Efficiency: " << avg_tflops << " TFLOPS\n\n\n" << std::endl;

    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_o);

    delete[] q, k, v, o, o_ref;
    delete[] q_bf, k_bf, v_bf, o_bf;

    return 0;
}
