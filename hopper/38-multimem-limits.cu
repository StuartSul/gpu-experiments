#include "hip/hip_runtime.h"
#include "multi-gpu.cuh"

constexpr int NUM_DEVICES = 8;

__global__ void all_reduce_float32_sum(float *data, int nelem);

int run(int verbose = 0) {

    assert(NUM_DEVICES > 1);

    /*
        Set up MC
    */
    CUCHECK(hipInit(0));

    size_t granularity;
    size_t size;

    hipMemGenericAllocationHandle_t mcHandle;
    CUmulticastObjectProp mcProp = {};
    mcProp.numDevices = NUM_DEVICES;
    mcProp.handleTypes = hipMemHandleTypePosixFileDescriptor; // single node
    mcProp.flags = 0; // SBZ

    granularity = 0;
    CUCHECK(cuMulticastGetGranularity(&granularity, &mcProp, CU_MULTICAST_GRANULARITY_RECOMMENDED)); 
    size = granularity;
    mcProp.size = size;
    CUCHECK(cuMulticastCreate(&mcHandle, &mcProp));

    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        hipDevice_t dev;
        CUCHECK(hipDeviceGet(&dev, dev_idx));
        CUCHECK(cuMulticastAddDevice(mcHandle, dev));
    }

    hipMemGenericAllocationHandle_t memHandles[NUM_DEVICES];
    hipDeviceptr_t mcPtrs[NUM_DEVICES];
    hipDeviceptr_t memPtrs[NUM_DEVICES];

    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));

        hipMemAllocationProp memProp = {};
        memProp.type = hipMemAllocationTypePinned;
        memProp.requestedHandleTypes = hipMemHandleTypePosixFileDescriptor;
        memProp.location.id = dev_idx;
        memProp.location.type = hipMemLocationTypeDevice;

        size_t mem_granularity = 0;
        CUCHECK(hipMemGetAllocationGranularity(&mem_granularity, &memProp, hipMemAllocationGranularityRecommended));
        if (size % mem_granularity != 0) {
            fprintf(stderr, "Size must be a multiple of mem granularity\n");
            exit(1);
        }

        // Allocate physical memory on the device
        CUCHECK(hipMemCreate(&memHandles[dev_idx], size, &memProp, 0));

        // Bind the physical memory to the multicast handle
        CUCHECK(cuMulticastBindMem(mcHandle, /*mcOffset=*/0, memHandles[dev_idx], /*memOffset=*/0, size, 0));
        
        // Allocate virtual address space for the handles
        CUCHECK(hipMemAddressReserve(&mcPtrs[dev_idx], size, granularity, 0, 0));
        CUCHECK(hipMemAddressReserve(&memPtrs[dev_idx], size, granularity, 0, 0));

        // Bind VAs to the multicast handle and physical memory
        CUCHECK(hipMemMap(mcPtrs[dev_idx], size, 0, mcHandle, 0));
        CUCHECK(hipMemMap(memPtrs[dev_idx], size, 0, memHandles[dev_idx], 0));

        // Remember to set access AFTER mapping
        hipMemAccessDesc desc[1];
        desc[0].flags = hipMemAccessFlagsProtReadWrite;
        desc[0].location.id = dev_idx;
        desc[0].location.type = hipMemLocationTypeDevice;
        CUCHECK(hipMemSetAccess(mcPtrs[dev_idx], size, desc, 1));
        CUCHECK(hipMemSetAccess(memPtrs[dev_idx], size, desc, 1));
    }

    /*
        Setup the data
    */
    assert(size % sizeof(float) == 0);

    int nelem = size / sizeof(float);
    float **host_mats = (float**)malloc(NUM_DEVICES * sizeof(float*));
    srand(static_cast<unsigned int>(time(nullptr))); // random seed

    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        host_mats[dev_idx] = (float*)malloc(size);
        if (verbose) printf("Device %d: ", dev_idx);
        for (int i = 0; i < nelem; ++i) {
            host_mats[dev_idx][i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
            if (i < 10)
                if (verbose) printf("%f ", host_mats[dev_idx][i]);
        }
        if (verbose) printf("... (%d elements)\n", nelem);
    }

    float *expected = (float*)malloc(size);
    if (verbose) printf("Expected: ");
    for (int i = 0; i < nelem; ++i) {
        expected[i] = 0.0f;
        for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
            expected[i] += host_mats[dev_idx][i];
        }
        if (i < 10)
            if (verbose) printf("%f ", expected[i]);
    }
    if (verbose) printf("... (%d elements)\n", nelem);

    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        CUDACHECK(hipMemcpy((void*)memPtrs[dev_idx], host_mats[dev_idx], size, hipMemcpyHostToDevice));
    }

    /*
        Perform the reduction
    */
    CUDACHECK(hipSetDevice(0));
    all_reduce_float32_sum<<<(nelem / 4 + 255) / 256, 256>>>((float*)mcPtrs[0], nelem);
    CUDACHECK(hipDeviceSynchronize());

    /* 
        Bring back data
    */
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        CUDACHECK(hipMemcpy(host_mats[dev_idx], (void*)memPtrs[dev_idx], size, hipMemcpyDeviceToHost));
    }

    /*
        Verify the results
    */
    float TOL = 1e-5;
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        if (verbose) printf("Device %d: ", dev_idx);
        for (int i = 0; i < nelem; ++i) {
            if (i < 10)
                if (verbose) printf("%f ", host_mats[dev_idx][i]);
            if (fabs(expected[i] - host_mats[dev_idx][i]) > TOL) {
                fprintf(stderr, "Mismatch at device %d, index %d: expected %f, got %f\n", dev_idx, i, expected[i], host_mats[dev_idx][i]);
                exit(1);
            }
        }
        if (verbose) printf("... (%d elements)\n", nelem);
    }

    /*
        Cleanup and exit
    */

    // Free resources
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));

        // Always free the memory in this order
        CUCHECK(hipMemUnmap(mcPtrs[dev_idx], size));
        CUCHECK(hipMemUnmap(memPtrs[dev_idx], size));
        CUCHECK(hipMemAddressFree(mcPtrs[dev_idx], size));
        CUCHECK(hipMemAddressFree(memPtrs[dev_idx], size));
        CUCHECK(hipMemRelease(memHandles[dev_idx]));
    }

    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        free(host_mats[dev_idx]);
    }

    free(host_mats);
    free(expected);

    if (verbose) printf("Done\n");
    return 0;
}

int main() {
    run(1); // warmup

    for (int i = 2; i <= 130; ++i) {
        printf("Run %d\n", i);
        run();
    }
}

__global__ void all_reduce_float32_sum(float *data, int nelem) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx * 4 < nelem) {
        volatile float4 val;
        float *ptr = data + idx * 4;
        asm volatile("multimem.ld_reduce.relaxed.sys.global.add.v4.f32 {%0, %1, %2, %3}, [%4];" : "=f"(val.x), "=f"(val.y), "=f"(val.z), "=f"(val.w) : "l"(ptr) : "memory"); // relaxed vs weak?
        asm volatile("fence.proxy.alias;" ::: "memory"); // force memory ordering
        // *ptr = val;
        asm volatile("multimem.st.relaxed.sys.global.v4.f32 [%0], {%1, %2, %3, %4};" :: "l"(ptr), "f"(val.x), "f"(val.y), "f"(val.z), "f"(val.w) : "memory"); // curious: what if I don't use asm here and just use plain assignment?
    }
}
