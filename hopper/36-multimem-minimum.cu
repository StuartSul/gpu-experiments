#include "hip/hip_runtime.h"
#include "multi-gpu.cuh"

using bf16 = __hip_bfloat16;

__global__ void writeKernel(bf16 *ptr, int nelem) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nelem)
        ptr[idx] = __int2bfloat16_rd(idx);
}

__global__ void readKernel(bf16 *ptr, int nelem) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 100)
        printf("ptr[%d] = %f ", idx, __bfloat162float(ptr[idx]));
}

int main() {

    // hipInit must be called before any Driver API calls, and argument SBZ
    CUCHECK(hipInit(0));

    // A generic allocation handle representing a multicast object
    hipMemGenericAllocationHandle_t mcHandle;

    // Describe the allocation handle
    CUmulticastObjectProp mcProp;
    mcProp.flags = 0;
    mcProp.handleTypes = 0;
    mcProp.numDevices = 2;

    size_t granularity;
    CUCHECK(cuMulticastGetGranularity(&granularity, &mcProp, CU_MULTICAST_GRANULARITY_RECOMMENDED));
    
    size_t size = 64 * 1024 * 1024;
    mcProp.size = size;
    size_t nelem = size / sizeof(bf16);
    
    CUCHECK(cuMulticastCreate(&mcHandle, &mcProp));
    CUCHECK(cuMulticastAddDevice(mcHandle, /*dev=*/0));
    CUCHECK(cuMulticastAddDevice(mcHandle, /*dev=*/1));

    bf16 *dev0ptr;
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipMalloc(&dev0ptr, size));
    CUCHECK(cuMulticastBindAddr(mcHandle, 0, (hipDeviceptr_t)dev0ptr, size, 0));

    bf16 *dev1ptr;
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMalloc(&dev1ptr, size));
    CUCHECK(cuMulticastBindAddr(mcHandle, 0, (hipDeviceptr_t)dev1ptr, size, 0));

    CUDACHECK(hipSetDevice(0));
    hipDeviceptr_t vaPtr0;
    CUCHECK(hipMemAddressReserve(&vaPtr0, size, granularity, 0, 0));
    CUCHECK(hipMemMap(vaPtr0, size, 0, mcHandle, 0));
    hipMemAccessDesc desc0[1];
    desc0[0].flags = hipMemAccessFlagsProtReadWrite;
    desc0[0].location.id = 0; /* device ID */
    desc0[0].location.type = hipMemLocationTypeDevice;
    CUCHECK(hipMemSetAccess(vaPtr0, size, desc0, 1));

    CUDACHECK(hipSetDevice(1));
    hipDeviceptr_t vaPtr1;
    CUCHECK(hipMemAddressReserve(&vaPtr1, size, granularity, 0, 0));
    CUCHECK(hipMemMap(vaPtr1, size, 0, mcHandle, 0));
    hipMemAccessDesc desc1[1];
    desc1[0].flags = hipMemAccessFlagsProtReadWrite;
    desc1[0].location.id = 1; /* device ID */
    desc1[0].location.type = hipMemLocationTypeDevice;
    CUCHECK(hipMemSetAccess(vaPtr1, size, desc1, 1));

    CUDACHECK(hipSetDevice(0));
    writeKernel<<<nelem / 256, 256>>>((bf16*)vaPtr0, nelem);
    CUDACHECK(hipDeviceSynchronize());

    CUDACHECK(hipSetDevice(1));
    readKernel<<<nelem / 256, 256>>>((bf16*)dev1ptr, nelem);
    CUDACHECK(hipDeviceSynchronize());

    printf("\nDone\n");    
    return 0;
}