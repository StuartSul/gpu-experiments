#include "hip/hip_runtime.h"
/*

    Nothing useful. Just so I can understand TK better

    Original output, run from TK directory:
    --------------------  M=4096 N=4096 K=4096  --------------------
    Block size: 128x256
    Allocated host memory
    Initialized matrices
    Performed CPU matrix multiplication
    Allocated device memory
    Copied matrices to device
    Launching warmup kernel with grid (132, 1), block (384)
    Launching kernel with grid (132, 1), block (384)
    Avg Kernel execution time: 176.418 us
    Achieved performance: 779.055 TFLOPs
    Copied result back to host
    Converted result back to float
    Max error: 0.0982647
    Error count: 0

    My output (previous 32 program):
    ------------------------- Benchmark -------------------------
    M = 4096, N = 4096, K = 4096
    Block size: 128x256

    Matrix A (M x K): 0.296543 -0.316565 0.279691 0.0968502 -0.0541672 -0.400025 -0.0407511 -0.166291 -0.357133 0.150888 
    Matrix B (K x N): 0.0904346 0.178184 0.267407 0.0267467 -0.389856 0.35117 -0.260957 -0.339091 -0.398018 0.262312 
    Expected C (M x N): -1.39006 1.66512 2.44395 -8.22833 2.22921 6.12578 -7.62248 -1.25781 3.26331 3.74322 

    Launching kernel with grid (132, 1), block (384)
        Execution time: 0.176627 ms
        Performance: 778.131 TFLOPs
    Matrix C (M x N): -1.38281 1.67188 2.45312 -8.25 2.23438 6.125 -7.625 -1.26562 3.28125 3.76562 
        Maximum error: 0.0982647
        Error count: 0
    -------------------------------------------------------------

    My output (TK kernel + all_reduce):
    ------------------------- Benchmark -------------------------
    M = 4096, N = 4096, K = 4096
    Block size: 128x256

    Matrix A (M x K): 0.296543 -0.316565 0.279691 0.0968502 -0.0541672 -0.400025 -0.0407511 -0.166291 -0.357133 0.150888 
    Matrix B (K x N): 0.0904346 0.178184 0.267407 0.0267467 -0.389856 0.35117 -0.260957 -0.339091 -0.398018 0.262312 
    Expected C (M x N): -1.39006 1.66512 2.44395 -8.22833 2.22921 6.12578 -7.62248 -1.25781 3.26331 3.74322 

    Launching kernels with grid (132, 1), block (384) on all devices
        Execution time: 0.191944 ms
    Matrix C (M x N): -1.39062 1.65625 2.4375 -8.1875 2.21875 6.125 -7.625 -1.24219 3.25 3.78125 
        Maximum error: 0.272608
        Error count: 0
    -------------------------------------------------------------

*/

#include <iostream>
#include <random>

#include "multi-gpu.cuh"

#include "kittens.cuh"
#include "prototype.cuh"
#include <hip/hip_bf16.h>

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcf;

constexpr int N = 40960;
constexpr int NUM_ITERS = 10; // number of iterations for benchmarking
constexpr int NUM_DEVICES = 8; // number of GPUs
constexpr int WARPSIZE = 32;
constexpr int STRIDE = 4;

template<int M_BLOCK, int N_BLOCK>
struct matmul_layout {
    using  base_tile      = st_bf<64, 64>;
    using  global_layout  = gl<bf16, 1, 1, -1, -1, base_tile>;
    struct globals        { global_layout A, B, C; };
    struct input_block    { base_tile a[M_BLOCK], b[N_BLOCK]; };
    struct finish_block   { base_tile c[M_BLOCK][N_BLOCK]; };
    struct common_state   { int2 coord; };
    struct consumer_state { rt_fl<16, N_BLOCK*base_tile::cols> accum; };
};

template<int _M_BLOCK=2, int _N_BLOCK=4, int _SUPER_M=12>
struct matmul_template {
    static constexpr int M_BLOCK = _M_BLOCK, N_BLOCK = _N_BLOCK, SUPER_M = _SUPER_M;
    using layout    = matmul_layout<M_BLOCK, N_BLOCK>;
    using wide_tile = st_bf<64, 64*N_BLOCK>;
    static constexpr int NUM_CONSUMER_WARPS=M_BLOCK*4, INPUT_PIPE_STAGES=4, PRODUCER_BARRIER_ARRIVALS=1;
    // Helper functions
    template<bool PERISISTENT_GRID=true> __host__ static inline dim3 grid(int M, int N, int K) {
        return dim3(PERISISTENT_GRID ? 132 : M*N/(M_BLOCK*N_BLOCK*layout::base_tile::num_elements));
    }
      // ThunderKittens template functions
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        int Rblocks = args.globals.C.rows / (M_BLOCK*64), Cblocks = args.globals.C.cols / (N_BLOCK*64);
        int super_rows = (Rblocks/SUPER_M)*SUPER_M,
            final_rows = Rblocks - super_rows,
            super_repeat = SUPER_M*Cblocks;
        int task_id = args.task_iter*gridDim.x + blockIdx.x;
        if (task_id < super_rows * Cblocks)
            args.common.coord = { SUPER_M*(task_id/super_repeat) + task_id%SUPER_M,
                           (task_id%super_repeat)/SUPER_M };
        else if (task_id < Rblocks*Cblocks) {
            int remainder_id = task_id - super_rows*Cblocks;
            args.common.coord = { super_rows + (remainder_id%final_rows), remainder_id/final_rows };
        }
        else { // Id is too high, no more work to do
            args.num_iters = -1;
            return;
        }
        args.num_iters = args.globals.A.cols/64;
        int id = warpgroup::groupid() == NUM_CONSUMER_WARPS/4 ? 0 : warpgroup::groupid(); // producer sets as 0
        args.common.coord = { args.common.coord.x*M_BLOCK + id, args.common.coord.y*N_BLOCK };
    }
    struct producer {
        __device__ static void setup(producer_setup_args<layout> args) {
            warpgroup::decrease_registers<40>(); // decrease registers for producers
        }
        __device__ static void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                for(int i = 0; i < M_BLOCK; i++)
                    tma::load_async(args.input.a[i], args.globals.A,
                                    {args.common.coord.x+i, args.iter}, args.inputs_arrived);
                for(int i = 0; i < N_BLOCK; i++)
                    tma::load_async(args.input.b[i], args.globals.B,
                                    {args.iter, args.common.coord.y+i}, args.inputs_arrived);
            }
        }
    };
    struct consumer {
        __device__ static void setup(consumer_setup_args<layout> args) {
            warpgroup::increase_registers<232>(); // increase registers for consumers
            zero(args.state.accum);
        }
        __device__ static void compute(consumer_compute_args<layout> args) {
            warpgroup::mma_AB(
                args.state.accum, // dest registers
                args.input.a[warpgroup::groupid()], // A matrix
                reinterpret_cast<wide_tile&>(args.input.b) // B matrix
            );
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.inputs_finished);
        }
        __device__ static void finish(consumer_finish_args<layout> args) {
            warpgroup::store(reinterpret_cast<wide_tile&>(args.finish.c[warpgroup::groupid()]), args.state.accum);
            warpgroup::sync(warpgroup::groupid()+4);
            if(warpgroup::warpid() == 0) for(int i = 0; i < N_BLOCK; i++) {
                tma::store_async(args.globals.C, args.finish.c[warpgroup::groupid()][i],
                                             {args.common.coord.x, args.common.coord.y+i});
                tma::store_async_read_wait(); // wait that store is finished before reusing finish memory
            }
            zero(args.state.accum);
            if(laneid() == 0) arrive(args.finish_finished);
        }
    };
};

__global__ void all_reduce_bf16(kittens::bf16 *device_mat, const int N);

template<typename mmt>
void inner_run(kittens::bf16 *device_A, kittens::bf16 *device_B, kittens::bf16 *device_C,
               size_t M, size_t N, size_t K, dim3 grid, dim3 block) {

    using global_layout = typename mmt::layout::global_layout;
    using globals = typename mmt::layout::globals;

    global_layout A_global{device_A, nullptr, nullptr, M, K};
    global_layout B_global{device_B, nullptr, nullptr, K, N};
    global_layout C_global{device_C, nullptr, nullptr, M, N};

    globals G{A_global, B_global, C_global};

    kittens::prototype::lcf::kernel<mmt><<<grid, block, MAX_SHARED_MEMORY - 1024>>>(G);
}

template<typename mmt>
void run(size_t M, size_t N, size_t K) {
    
    std::cout << "------------------------- Benchmark -------------------------\n";
    std::cout << "  M = " << M << ", N = " << N << ", K = " << K << "\n";
    std::cout << "  Block size: " << mmt::M_BLOCK * 64 << "x" << mmt::N_BLOCK * 64 << "\n";

    // Host-side matrices
    float *host_A = new float[M * K];
    float *host_B = new float[K * N];
    float *host_C = new float[M * N];
    float *host_C_ref = new float[M * N];

    // Initialize A & B matrices
    std::mt19937 prng(42);
    std::uniform_real_distribution<> random(-0.5, 0.5);
    std::cout << "\n  Matrix A (M x K): ";
    for (int i = 0; i < M * K; ++i) {
        host_A[i] = random(prng);
        if (i < 10)
            std::cout << host_A[i] << " ";
    }
    std::cout << "\n  Matrix B (K x N): ";
    for (int i = 0; i < K * N; ++i) {
        host_B[i] = random(prng);
        if (i < 10)
            std::cout << host_B[i] << " ";
    }
    std::cout << "\n";

    // Generate expected output (just do first 10x10 tile)
    std::cout << "  Expected C (M x N): ";
    #pragma omp parallel for collapse(2)
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += host_A[i * K + k] * host_B[k * N + j];
            }
            host_C_ref[i * N + j] = sum;
        }
    }
    for (int i = 0; i < 10; i++) {
        std::cout << host_C_ref[i] << " ";
    }
    std::cout << "\n";

    // Convert to BF16 on host
    __hip_bfloat16 *host_A_bf16 = new __hip_bfloat16[M * K];
    __hip_bfloat16 *host_B_bf16 = new __hip_bfloat16[K * N];
    for (int i = 0; i < M * K; ++i) host_A_bf16[i] = __float2bfloat16(host_A[i]);
    for (int i = 0; i < K * N; ++i) host_B_bf16[i] = __float2bfloat16(host_B[i]);

    // Allocate device-side matrices
    int K_sh = K / NUM_DEVICES;
    __hip_bfloat16 *device_A_sh[NUM_DEVICES], *device_B_sh[NUM_DEVICES]; // *device_C[NUM_DEVICES]
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        CUDACHECK(hipMalloc(&device_A_sh[dev_idx], M * K_sh * sizeof(__hip_bfloat16)));
        CUDACHECK(hipMalloc(&device_B_sh[dev_idx], K_sh * N * sizeof(__hip_bfloat16)));
        // CUDACHECK(hipMalloc(&device_C[dev_idx], M * N * sizeof(__hip_bfloat16)));
    }

    // Copy to device matrices
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        for (int i = 0; i < M; ++i) { // TODO: do a single hipMemcpy
            CUDACHECK(hipMemcpy(device_A_sh[dev_idx] + i * K_sh,      // i-th row of device A
                                 host_A_bf16 + i * K + dev_idx * K_sh, // i-th row, dev_idx-th block of host A
                                 K_sh * sizeof(__hip_bfloat16), 
                                 hipMemcpyHostToDevice));
        }
        // Since B is sharded row-wise, we can do a single hipMemcpy
        CUDACHECK(hipMemcpy(device_B_sh[dev_idx], 
                             host_B_bf16 + dev_idx * K_sh * N, 
                             K_sh * N * sizeof(__hip_bfloat16), 
                             hipMemcpyHostToDevice));
    }

    /*
        Setup multimem stuff
    */
    assert(NUM_DEVICES > 1);
    CUCHECK(hipInit(0));

    // Check multicast support
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        hipDevice_t dev;
        CUCHECK(hipDeviceGet(&dev, dev_idx));
        int deviceSupportsMultiCast;
        CUCHECK(hipDeviceGetAttribute(
            &deviceSupportsMultiCast, CU_DEVICE_ATTRIBUTE_MULTICAST_SUPPORTED, dev));
        if (!deviceSupportsMultiCast) {
            fprintf(stderr, "Device %d does not support Multicast Objects\n", dev_idx);
            exit(1);
        }
    }

    // Create multicast handle for matrix C
    CUmulticastObjectProp mcProp = {};
    hipMemGenericAllocationHandle_t mcHandle;
    mcProp.numDevices = NUM_DEVICES;
    mcProp.handleTypes = hipMemHandleTypePosixFileDescriptor; // single node
    mcProp.flags = 0; // SBZ
    mcProp.size = M * N * sizeof(__hip_bfloat16);
    CUCHECK(cuMulticastCreate(&mcHandle, &mcProp));

    // Add all devices to the multicast handle
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        hipDevice_t dev;
        CUCHECK(hipDeviceGet(&dev, dev_idx));
        CUCHECK(cuMulticastAddDevice(mcHandle, dev));
    }

    // Allocate, bind, and map matrix C on each device
    hipMemGenericAllocationHandle_t device_C_phymem[NUM_DEVICES];
    hipDeviceptr_t device_C[NUM_DEVICES];
    hipDeviceptr_t device_C_mc[NUM_DEVICES];
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        hipMemAllocationProp memProp = {};
        memProp.type = hipMemAllocationTypePinned;
        memProp.requestedHandleTypes = hipMemHandleTypePosixFileDescriptor;
        memProp.location.id = dev_idx;
        memProp.location.type = hipMemLocationTypeDevice;
        CUCHECK(hipMemCreate(&device_C_phymem[dev_idx], M * N * sizeof(__hip_bfloat16), &memProp, 0));
        CUCHECK(cuMulticastBindMem(mcHandle, 0, device_C_phymem[dev_idx], 0, M * N * sizeof(__hip_bfloat16), 0));
        CUCHECK(hipMemAddressReserve(&device_C[dev_idx], M * N * sizeof(__hip_bfloat16),  2 * 1024 * 1024, 0, 0)); // TODO: change alignment
        CUCHECK(hipMemAddressReserve(&device_C_mc[dev_idx], M * N * sizeof(__hip_bfloat16),  2 * 1024 * 1024, 0, 0)); // TODO: change alignment
        CUCHECK(hipMemMap(device_C[dev_idx], M * N * sizeof(__hip_bfloat16), 0, device_C_phymem[dev_idx], 0));
        CUCHECK(hipMemMap(device_C_mc[dev_idx], M * N * sizeof(__hip_bfloat16), 0, mcHandle, 0));
        hipMemAccessDesc desc[1];
        desc[0].flags = hipMemAccessFlagsProtReadWrite;
        desc[0].location.id = dev_idx;
        desc[0].location.type = hipMemLocationTypeDevice;
        CUCHECK(hipMemSetAccess(device_C[dev_idx], M * N * sizeof(__hip_bfloat16), desc, 1));
        CUCHECK(hipMemSetAccess(device_C_mc[dev_idx], M * N * sizeof(__hip_bfloat16), desc, 1));
    }

    // Prepare kernel launch
    ThreadGang gang(NUM_DEVICES); // threadpool
    gang.execute([](int dev_idx) { // set device on each thread
        CUDACHECK(hipSetDevice(dev_idx));
    });
    unsigned long smem_size = kittens::MAX_SHARED_MEMORY - 1024; // MAX_SHARED_MEMORY = 227KB for Hopper
    gang.execute([smem_size](int dev_idx) {
        CUDACHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(kittens::prototype::lcf::kernel<mmt>), 
                                       hipFuncAttributeMaxDynamicSharedMemorySize, 
                                       smem_size));
    });
    dim3 grid(mmt::grid(M, N, K_sh)); // use sharded K
    dim3 block(kittens::prototype::detail::NUM_THREADS_v<mmt>);
    for (int i = 0; i < 2; ++i) { // warmup
        gang.execute([&device_A_sh, &device_B_sh, &device_C, &M, &N, &K_sh, &grid, &block](int dev_idx) { // warmup
            inner_run<mmt>(device_A_sh[dev_idx], device_B_sh[dev_idx], (kittens::bf16 *)device_C[dev_idx], M, N, K_sh, grid, block);
            CUDACHECK(hipDeviceSynchronize());
        });
        gang.execute([&M, &N, &device_C_mc](int dev_id) { 
            int nelem = M * N;
            int nelem_per_dev = nelem / NUM_DEVICES;
            int offset = nelem_per_dev * dev_id;
            all_reduce_bf16<<<(nelem_per_dev + 2048 * STRIDE - 1) / (2048 * STRIDE), 256>>>((__hip_bfloat16 *)(device_C_mc[dev_id]) + offset, nelem_per_dev);
            do {                                   
                hipError_t err = hipDeviceSynchronize();                                    
                if (err != hipSuccess) {                                 
                    fprintf(stderr, "(dev %d) Failed: CUDA error %s:%d '%s'\n",    
                        dev_id, __FILE__, __LINE__, hipGetErrorString(err));     
                    exit(EXIT_FAILURE);
                }
            } while(0);
        });
    }
    
    // Start timing
    std::cout << "\n  Launching kernels with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ") on all devices\n";
    auto start = std::chrono::high_resolution_clock::now();

    // Launch!
    for (int i = 0; i < NUM_ITERS; ++i) {
        gang.execute([&device_A_sh, &device_B_sh, &device_C, &M, &N, &K_sh, &grid, &block](int dev_idx) {
            inner_run<mmt>(device_A_sh[dev_idx], device_B_sh[dev_idx], (kittens::bf16 *)device_C[dev_idx], M, N, K_sh, grid, block);
            CUDACHECK(hipDeviceSynchronize());
        });
        gang.execute([&M, &N, &device_C_mc](int dev_id) { 
            int nelem = M * N;
            int nelem_per_dev = nelem / NUM_DEVICES;
            int offset = nelem_per_dev * dev_id;
            all_reduce_bf16<<<(nelem_per_dev + 2048 * STRIDE - 1) / (2048 * STRIDE), 256>>>((__hip_bfloat16 *)(device_C_mc[dev_id]) + offset, nelem_per_dev);
        });
    }

    // End timing
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    double avg_seconds = elapsed.count() / NUM_ITERS;

    // Calculate results
    // double n_fp_ops = (double)2.0 * M * N * K; // 2 floating point ops per multiply-add
    // double tflops = (n_fp_ops / 1e12) / avg_seconds;
    std::cout << "    Execution time: " << (avg_seconds * 1e3) << " ms\n";
    // std::cout << "    Performance: " << tflops << " TFLOPs\n";

    // Copy & convert back to host
    __hip_bfloat16 *host_C_bf16 = new __hip_bfloat16[M * N];
    // for (int i = 0; i < M * N; ++i) host_C[i] = 0.0; // accumulate on host
    // for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
    //     CUDACHECK(hipSetDevice(dev_idx));
    //     CUDACHECK(hipMemcpy(host_C_bf16, (void *)device_C[dev_idx], M * N * sizeof(__hip_bfloat16), hipMemcpyDeviceToHost));
    //     for (int i = 0; i < M * N; ++i) host_C[i] += __bfloat162float(host_C_bf16[i]);
    // }

    // Should work on any device
    int random_dev_idx = 3;
    CUDACHECK(hipSetDevice(random_dev_idx));
    CUDACHECK(hipMemcpy(host_C_bf16, (void *)device_C_mc[random_dev_idx], M * N * sizeof(__hip_bfloat16), hipMemcpyDeviceToHost));
    for (int i = 0; i < M * N; ++i) host_C[i] = __bfloat162float(host_C_bf16[i]);

    std::cout << "  Matrix C (M x N): ";
    for (int i = 0; i < 10; i++) {
        std::cout << host_C[i] << " ";
    }
    std::cout << "\n";

    // Verify result (just do first 10x10 tile)
    float max_error = 0.f;
    int n_error = 0;
    for (int i = 0; i < 10; ++i) {
        for (int j = 0; j < 10; j++) {
            float error = std::abs(host_C[i * N + j] - host_C_ref[i * N + j]);
            if (error > 1.0) // large due to bf16 <-> fp32 conversion
                ++n_error;
            max_error = std::max(max_error, error);
        }
    }
    std::cout << "    Maximum error: " << max_error << "\n";
    std::cout << "    Error count (out of 10x10): " << n_error << "\n";
    std::cout << "-------------------------------------------------------------\n";

    // Clean up
    delete[] host_A;
    delete[] host_A_bf16;
    delete[] host_B;
    delete[] host_B_bf16;
    delete[] host_C;
    delete[] host_C_bf16;
    delete[] host_C_ref;
    CUDACHECK(hipFree(device_A_sh));
    CUDACHECK(hipFree(device_B_sh));
    // CUDACHECK(hipFree(device_C));

    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        CUCHECK(hipMemUnmap(device_C[dev_idx], M * N * sizeof(__hip_bfloat16)));
        CUCHECK(hipMemUnmap(device_C_mc[dev_idx], M * N * sizeof(__hip_bfloat16)));
        CUCHECK(hipMemAddressFree(device_C[dev_idx], M * N * sizeof(__hip_bfloat16)));
        CUCHECK(hipMemAddressFree(device_C_mc[dev_idx], M * N * sizeof(__hip_bfloat16)));
        CUCHECK(hipMemRelease(device_C_phymem[dev_idx]));
    }
}

int main() {
    run<matmul_template<2, 4, 8>>(N, N, N);
    return 0;
}

__global__ void all_reduce_bf16(kittens::bf16 *device_mat, const int N) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = tid / WARPSIZE;
    int lane_id = threadIdx.x % WARPSIZE;

    constexpr int nelem_per_iter = 16 / sizeof(__hip_bfloat16);
    constexpr int nelem_per_warp_per_iter = nelem_per_iter * WARPSIZE;
    constexpr int nelem_per_warp = STRIDE * nelem_per_warp_per_iter;
    int start_idx = nelem_per_warp * warp_id;

    for (int i = 0; i < STRIDE; ++i) {
        int idx = start_idx + i * nelem_per_warp_per_iter + lane_id * nelem_per_iter;
        if (idx < N) {
            volatile float x, y, z, w; // "float" used just to hold 2 bfloat16 values, and we can't pass __hip_bfloat16 directly to inline asm
            __hip_bfloat16 *ptr = (__hip_bfloat16 *)(device_mat + idx);
            // asm volatile("multimem.ld_reduce.relaxed.sys.global.add.v8.bf16 {%0, %1, %2, %3, %4, %5, %6, %7}, [%4];" : "=h"(x0), "=h"(x1), "=h"(x2), "=h"(x3), "=h"(x4), "=h"(x5), "=h"(x6), "=h"(x7) : "l"(ptr) : "memory");
            // asm volatile("multimem.st.relaxed.sys.global.v8.bf16 [%0], {%1, %2, %3, %4, %5, %6, %7, %8};" :: "l"(ptr), "h"(x0), "h"(x1), "h"(x2), "h"(x3), "h"(x4), "h"(x5), "h"(x6), "h"(x7) : "memory");
            asm volatile("multimem.ld_reduce.relaxed.sys.global.add.v4.bf16x2 {%0, %1, %2, %3}, [%4];" : "=f"(x), "=f"(y), "=f"(z), "=f"(w) : "l"(ptr) : "memory");
            asm volatile("multimem.st.relaxed.sys.global.v4.bf16x2 [%0], {%1, %2, %3, %4};" :: "l"(ptr), "f"(x), "f"(y), "f"(z), "f"(w) : "memory");
        }
        __syncthreads();
    }
}
