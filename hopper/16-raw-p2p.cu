#include "hip/hip_runtime.h"
#include "multi-gpu.cuh"

using namespace std;

__global__ void copyKernel(int* src, int* dst) {
    // Copy value from src to dst
    // *dst = *src;
    volatile int value;
    asm volatile (
        "{ ld.global.u32 %0, [%1];"
        "st.global.u32 [%2], %0; }"
        : "=r"(value)
        : "l"(src), "l"(dst)
        : "memory"
    );
}

int main() {

    // P2P Setup
    int can_access_peer_0_1;
    int can_access_peer_1_0;
    CUDACHECK(hipDeviceCanAccessPeer(&can_access_peer_0_1, 0, 1));
    CUDACHECK(hipDeviceCanAccessPeer(&can_access_peer_1_0, 1, 0));
    cout << "Device 0 can access device 1: " << can_access_peer_0_1 << endl;
    cout << "Device 1 can access device 0: " << can_access_peer_1_0 << endl;
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipDeviceEnablePeerAccess(1, 0));
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipDeviceEnablePeerAccess(0, 0));

    // Step 1: Allocate device memory (two separate locations, each 4 bytes)
    int *d_src, *d_dst;
    
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipMalloc((void**)&d_src, sizeof(int)));

    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMalloc((void**)&d_dst, sizeof(int)));

    // Step 2: Copy an integer from host to device (d_src)
    int h_value = 0xdeadbeef;
    CUDACHECK(hipSetDevice(0)); // since 0 has the source, this becomes p2p write
    CUDACHECK(hipMemcpy(d_src, &h_value, sizeof(int), hipMemcpyHostToDevice));
    
    // Step 3: Launch the kernel to copy value from d_src to d_dst
    CUDACHECK(hipSetDevice(0)); // since 0 has the source, this becomes p2p write
    copyKernel<<<1, 1>>>(d_src, d_dst);
    CUDACHECK(hipDeviceSynchronize());  // Ensure kernel execution is completed

    // Step 4: Copy d_dst back to the host
    int h_result;
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMemcpy(&h_result, d_dst, sizeof(int), hipMemcpyDeviceToHost));

    // Step 5: Verify correctness
    if (h_value == h_result) {
        std::cout << "Success! Value correctly copied: " << h_result << std::endl;
    } else {
        std::cerr << "Error! Mismatch: expected " << h_value << ", but got " << h_result << std::endl;
    }

    // Cleanup
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipFree(d_src));
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipFree(d_dst));

    return 0;
}
