#include "hip/hip_runtime.h"
/*
    Transfer time: 18.473 ms
    Bandwidth: 541.34 GB/s
*/

#include "kittens.cuh"

// Kernel to initialize memory with a value
__global__ void initKernel(float* data, float value, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
    for (size_t i = idx; i < n; i += stride) {
        data[i] = value;
    }
}

// Kernel to copy data from one GPU to another with coalesced access
__global__ void copyKernel(float* dst, const float* src, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
    // Coalesced access pattern: consecutive threads access consecutive memory locations
    for (size_t i = idx; i < n; i += stride) {
        dst[i] = src[i];
    }
}

// Kernel to verify data correctness
__global__ void verifyKernel(float* data, float expected, size_t n, int* errorCount) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
    for (size_t i = idx; i < n; i += stride) {
        if (fabsf(data[i] - expected) > 1e-5f) {
            atomicAdd(errorCount, 1);
        }
    }
}

int main() {
    // Configuration
    const size_t dataSize = 10ULL * 1024 * 1024 * 1024;  // 10 GB
    const size_t numElements = dataSize / sizeof(float);
    const float srcValue = 3.14f;
    const float dstInitValue = 0.0f;
    
    printf("NVLink Bandwidth Test (Kernel-based Copy)\n");
    printf("==========================================\n");
    printf("Data size: %.2f GB\n", dataSize / (1024.0 * 1024.0 * 1024.0));
    printf("Number of float elements: %zu\n", numElements);
    
    // Allocate memory on device 0
    float* d0_data;
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipMalloc(&d0_data, dataSize));
    printf("Allocated %.2f GB on Device 0\n", dataSize / (1024.0 * 1024.0 * 1024.0));

    // Initialize device 0 memory with 3.14
    int blockSize = 256;
    int gridSize = (numElements + blockSize - 1) / blockSize;
    gridSize = min(gridSize, 65536);

    initKernel<<<gridSize, blockSize>>>(d0_data, srcValue, numElements);
    CUDACHECK(hipDeviceSynchronize());
    printf("Initialized Device 0 memory with value: %.2f\n", srcValue);
    
    // Allocate memory on device 1
    float* d1_data;
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMalloc(&d1_data, dataSize));
    printf("Allocated %.2f GB on Device 1\n", dataSize / (1024.0 * 1024.0 * 1024.0));
    
    // Initialize device 1 memory with 0
    initKernel<<<gridSize, blockSize>>>(d1_data, dstInitValue, numElements);
    CUDACHECK(hipDeviceSynchronize());
    printf("Initialized Device 1 memory with value: %.2f\n\n", dstInitValue);
    
    // Enable peer access
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipDeviceEnablePeerAccess(1, 0));
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipDeviceEnablePeerAccess(0, 0));
    
    // Create events for timing
    CUDACHECK(hipSetDevice(0));
    hipEvent_t start, stop;
    CUDACHECK(hipEventCreate(&start));
    CUDACHECK(hipEventCreate(&stop));
    
    // Warm up run
    printf("Performing warm-up transfer...\n");
    copyKernel<<<gridSize, blockSize>>>(d1_data, d0_data, numElements);
    CUDACHECK(hipDeviceSynchronize());
    
    // Re-initialize device 1 for accurate test
    CUDACHECK(hipSetDevice(1));
    initKernel<<<gridSize, blockSize>>>(d1_data, dstInitValue, numElements);
    CUDACHECK(hipDeviceSynchronize());
    
    // Timed kernel copy
    printf("\nStarting timed kernel transfer: Device 0 -> Device 1\n");
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipEventRecord(start));
    
    copyKernel<<<gridSize, blockSize>>>(d1_data, d0_data, numElements);
    
    CUDACHECK(hipEventRecord(stop));
    CUDACHECK(hipDeviceSynchronize());
    
    float milliseconds = 0;
    CUDACHECK(hipEventElapsedTime(&milliseconds, start, stop));
    
    double seconds = milliseconds / 1000.0;
    double gigabytes = dataSize / (1024.0 * 1024.0 * 1024.0);
    double bandwidth_GBps = gigabytes / seconds;
    
    printf("\nTransfer Results:\n");
    printf("-----------------\n");
    printf("Transfer time: %.3f ms\n", milliseconds);
    printf("Bandwidth: %.2f GB/s\n", bandwidth_GBps);
    
    // Verify correctness on device 1
    printf("\nVerifying data correctness on Device 1...\n");
    CUDACHECK(hipSetDevice(1));
    
    int* d_errorCount;
    CUDACHECK(hipMalloc(&d_errorCount, sizeof(int)));
    CUDACHECK(hipMemset(d_errorCount, 0, sizeof(int)));
    
    verifyKernel<<<gridSize, blockSize>>>(d1_data, srcValue, numElements, d_errorCount);
    CUDACHECK(hipDeviceSynchronize());
    
    int h_errorCount;
    CUDACHECK(hipMemcpy(&h_errorCount, d_errorCount, sizeof(int), hipMemcpyDeviceToHost));
    
    if (h_errorCount == 0) {
        printf("✓ Correctness check PASSED: All values match expected value (%.2f)\n", srcValue);
    } else {
        printf("✗ Correctness check FAILED: %d mismatches found\n", h_errorCount);
    }
    
    // Sample a few values for verification
    float sample[10];
    CUDACHECK(hipMemcpy(sample, d1_data, sizeof(sample), hipMemcpyDeviceToHost));
    printf("\nFirst 10 values on Device 1 after transfer: ");
    for (int i = 0; i < 10; i++) {
        printf("%.2f ", sample[i]);
    }
    printf("\n");
    
    // Cleanup
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipFree(d0_data));
    CUDACHECK(hipEventDestroy(start));
    CUDACHECK(hipEventDestroy(stop));
    
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipFree(d1_data));
    CUDACHECK(hipFree(d_errorCount));
    
    printf("\nTest completed successfully!\n");
    
    return 0;
}