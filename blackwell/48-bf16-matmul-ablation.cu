#include "hip/hip_runtime.h"
/*
    Benchmarks on 16384x16384x16384 2-CTA matmuls.
    Note that the very first matmul (512x64x256) cannot be run by changing params. Code has to be reverted for that.

    - 512x64x256, 4-stage: 1570 TFLOP/s (running two 256x64x256 matmuls at a time)
    - 256x64x256, 4-stage: 1545 TFLOP/s (from here, running one 256x64x256 matmuls at a time)
    - 256x128x256, 2-stage: 1330 TFLOP/s
    - 256x128x256, 3-stage: 1513 TFLOp/s
    - 128x64x256,  4-stage: 1062 TFLOp/s
    - 128x128x256, 3-stage: 1200 TFLOP/s
    - 128x128x256, 4-stage: 1309 TFLOp/s
    - 256x64x128,  4-stage: 1066 TFLOp/s
    - 256x128x128, 3-stage: 1235 TFLOp/s
    - 256x128x128, 4-stage: 1340 TFLOp/s
    - 128x64x128,  4-stage: 619 TFLOp/s
    - 128x64x128,  5-stage: 653 TFLOp/s
    - 128x128x128, 4-stage: 902 TFLOp/s
    - 128x128x128, 5-stage: 883 TFLOp/s

    Deepseek shapes:
    - 256x192x128, 2-stage: 968 TFLOP/s
    - 256x192x128, 3-stage: 1354.10 TFLOp/s  <-- Wow!

    Observations:
      - chopping C_smem into pieces do not affect perf, only gives us more SMEM!
      - number of stages does matter a lot. it usually peaks at 4
*/

#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;

struct config {
    static constexpr int CLUSTER_SIZE = 2;
    static constexpr int SM_COUNT = 148;
    static constexpr int STATIC_SHARED_MEMORY = 128;
    static constexpr int DYNAMIC_SHARED_MEMORY = MAX_SHARED_MEMORY - STATIC_SHARED_MEMORY;

    static constexpr int CONSUMER_WARPGROUPS = 1;
    static constexpr int PRODUCER_WARPGROUPS = 1;
    static constexpr int NUM_WARPGROUPS = CONSUMER_WARPGROUPS + PRODUCER_WARPGROUPS;
    static constexpr int NUM_WARPS = NUM_WARPGROUPS * WARPGROUP_WARPS;
    static constexpr int NUM_THREADS = NUM_WARPS * WARP_THREADS;

    static constexpr int PRODUCER_REGISTERS = 56;
    static constexpr int CONSUMER_REGISTERS = 224;

    static constexpr int PIPELINE_STAGES = 3;
};

struct globals {
    static constexpr int SUPERGROUP_BLOCKS = 8;
    static constexpr int ROW_BLOCK = 256;
    static constexpr int COL_BLOCK = 128;
    static constexpr int REDUCTION_BLOCK = 192;

    using A_tile = st_bf<ROW_BLOCK / 2, REDUCTION_BLOCK>; // cluster distributed
    using B_tile = st_bf<COL_BLOCK / 2, REDUCTION_BLOCK>; // cluster distributed
    using C_tile = st_bf<ROW_BLOCK / 2, COL_BLOCK / 8>;   // cluster distributed

    gl<bf16, 1, 1, -1, -1, A_tile> A;
    gl<bf16, 1, 1, -1, -1, B_tile> B;
    gl<bf16, 1, 1, -1, -1, C_tile> C;

    __host__ inline dim3 grid() { return dim3(config::SM_COUNT); }
    __host__ inline dim3 block() { return dim3(config::NUM_THREADS); }
    __host__ inline int dynamic_shared_memory() { return config::DYNAMIC_SHARED_MEMORY; }

    struct pipeline_inputs {
        A_tile A;
        B_tile B;
    };

    struct pipeline_outputs {
        C_tile C;
    };
};

__global__ __cluster_dims__(config::CLUSTER_SIZE) __launch_bounds__(config::NUM_THREADS, 1)
void kernel(const __grid_constant__ globals G) {
    // Shared memory declaration
    extern __shared__ int __shm[];
    tma_swizzle_allocator allocator((int*)&__shm[0]);

    // Warpgroup configuration
    using consumer = group<config::CONSUMER_WARPGROUPS * WARPGROUP_WARPS>;
    int warpgroup_id = warpgroup::groupid();
    int warp_id = warpgroup::warpid();
    int lane_id = warp::laneid();

    // Allocate shared and tensor memory
    static_assert(sizeof(globals::pipeline_inputs) * config::PIPELINE_STAGES + sizeof(globals::pipeline_outputs) <= config::DYNAMIC_SHARED_MEMORY);
    globals::pipeline_inputs (&inputs)[config::PIPELINE_STAGES] = allocator.allocate<globals::pipeline_inputs, config::PIPELINE_STAGES>();
    globals::pipeline_outputs &outputs = allocator.allocate<globals::pipeline_outputs>();
    tensor_allocator<1, 2> tm_allocator {};

    // Set up mbarriers
    __shared__ semaphore inputs_arrived[config::PIPELINE_STAGES];
    __shared__ semaphore inputs_finished[config::PIPELINE_STAGES];
    __shared__ semaphore outputs_arrived;
    __shared__ semaphore tensors_finished;
    if (threadIdx.x == 0) {
        for (int i = 0; i < config::PIPELINE_STAGES; ++i) {
            init_semaphore(inputs_arrived[i], 0, 2);
            init_semaphore(inputs_finished[i], 0, 1);
        }
        init_semaphore(outputs_arrived, 0, 1);
        init_semaphore(tensors_finished, 0, 2);
    }
    everyone::tma::cluster::sync();

    // Pipeline configuration
    int num_blocks_per_row = G.C.cols() / globals::COL_BLOCK;
    int num_blocks_per_col = G.C.rows() / globals::ROW_BLOCK;
    int num_blocks = num_blocks_per_row * num_blocks_per_col;
    int num_iters_per_block = G.A.cols() / globals::REDUCTION_BLOCK;
    int num_blocks_per_supergroup = globals::SUPERGROUP_BLOCKS * num_blocks_per_row;

    // Declare stage and phasebits for semaphore waits
    int stage = 0;
    int last_stage = -1;
    uint32_t phasebits = 0xFFFF0000;

    // Main divergence
    if (warpgroup_id == config::NUM_WARPGROUPS - 1) {
        // Producer group
        warpgroup::decrease_registers<config::PRODUCER_REGISTERS>();
        int ctarank = cluster_ctarank();

        // Sub divergence
        if (warp_id == 3 && lane_id == 0) {
            // Producer group -- loaders
            for (int block_idx = clusterIdx().x; block_idx < num_blocks; block_idx += gridDim.x / config::CLUSTER_SIZE) {
                // Compute block indices
                int supergroup_idx = block_idx / num_blocks_per_supergroup;
                int idx_within_supergroup = block_idx % num_blocks_per_supergroup;
                int rows_in_supergroup = min(globals::SUPERGROUP_BLOCKS, num_blocks_per_col - supergroup_idx * globals::SUPERGROUP_BLOCKS);
                int row_within_supergroup = idx_within_supergroup % rows_in_supergroup;
                int row_block_idx = supergroup_idx * globals::SUPERGROUP_BLOCKS + row_within_supergroup;
                int col_block_idx = idx_within_supergroup / rows_in_supergroup;

                for (int i = 0; i < num_iters_per_block; ++i) {
                    tma::cluster::wait(inputs_finished[stage], get_phasebit<1>(phasebits, stage));
                    if (stage == last_stage) {
                        arrive(outputs_arrived);
                        last_stage = -1;
                    }
                    tma::cluster::expect_bytes(inputs_arrived[stage], sizeof(globals::pipeline_inputs), 0);
                    tma::cluster::load_async(inputs[stage].A, G.A, {row_block_idx * 2 + ctarank, i}, inputs_arrived[stage], (uint16_t)(1 << ctarank), 0);
                    tma::cluster::load_async(inputs[stage].B, G.B, {col_block_idx * 2 + ctarank, i}, inputs_arrived[stage], (uint16_t)(1 << ctarank), 0);
                    update_phasebit<1>(phasebits, stage);
                    if (i == num_iters_per_block - 1) {
                        last_stage = stage;
                    }
                    stage = (stage + 1) % config::PIPELINE_STAGES;
                }
            }
            tma::cluster::wait(inputs_finished[last_stage], get_phasebit<1>(phasebits, last_stage));
            arrive(outputs_arrived);
        } else if (lane_id == 0 && ctarank == 0 && warp_id == 0) {
            // Producer group -- launchers
            auto tm = tm_allocator.allocate<tt<float, globals::ROW_BLOCK / 2, globals::COL_BLOCK>>(0);
            for (int block_idx = clusterIdx().x; block_idx < num_blocks; block_idx += gridDim.x / config::CLUSTER_SIZE) {
                tma::cluster::wait(tensors_finished, get_phasebit<1>(phasebits, config::PIPELINE_STAGES));
                update_phasebit<1>(phasebits, config::PIPELINE_STAGES);
                {
                    tma::cluster::wait(inputs_arrived[stage], get_phasebit<0>(phasebits, stage));
                    mm2_ABt(tm, inputs[stage].A, inputs[stage].B, inputs_finished[stage]);
                    update_phasebit<0>(phasebits, stage);
                    stage = (stage + 1) % config::PIPELINE_STAGES;
                }
                for (int i = 1; i < num_iters_per_block; ++i) {
                    tma::cluster::wait(inputs_arrived[stage], get_phasebit<0>(phasebits, stage));
                    mma2_ABt(tm, inputs[stage].A, inputs[stage].B, inputs_finished[stage]);
                    update_phasebit<0>(phasebits, stage);
                    stage = (stage + 1) % config::PIPELINE_STAGES;
                }
            }
        }
    } else {
        // Consumer group
        warpgroup::increase_registers<config::CONSUMER_REGISTERS>();
        int ctarank = cluster_ctarank();
        auto tm = tm_allocator.allocate<tt<float, globals::ROW_BLOCK / 2, globals::COL_BLOCK>>(0);

        for (int block_idx = clusterIdx().x; block_idx < num_blocks; block_idx += gridDim.x / config::CLUSTER_SIZE) {
            // Compute block indices
            int supergroup_idx = block_idx / num_blocks_per_supergroup;
            int idx_within_supergroup = block_idx % num_blocks_per_supergroup;
            int rows_in_supergroup = min(globals::SUPERGROUP_BLOCKS, num_blocks_per_col - supergroup_idx * globals::SUPERGROUP_BLOCKS);
            int row_within_supergroup = idx_within_supergroup % rows_in_supergroup;
            int row_block_idx = supergroup_idx * globals::SUPERGROUP_BLOCKS + row_within_supergroup;
            int col_block_idx = idx_within_supergroup / rows_in_supergroup;

            // Wait for the last matmul to complete
            wait(outputs_arrived, get_phasebit<0>(phasebits, config::PIPELINE_STAGES));
            update_phasebit<0>(phasebits, config::PIPELINE_STAGES);

            // Load the output from tensor memory into registers
            rt_bf<globals::ROW_BLOCK / 8, globals::COL_BLOCK / 8> C[8];
            #pragma unroll
            for (int i = 0; i < 8; i++)
                consumer::load_async(C[i], tm.subtile<tt<float, globals::ROW_BLOCK / 2, globals::COL_BLOCK / 8>>(0, i * globals::COL_BLOCK / 8));
            tensor_load_wait();
            consumer::sync(1);
            if (consumer::laneid() == 0)
                tma::cluster::arrive(tensors_finished, 0);

            // Store to global memory
            #pragma unroll
            for (int i = 0; i < 8; i++) {
                consumer::store(outputs.C, C[i]);
                consumer::sync(1);
                consumer::tma::store_async(G.C, outputs.C, {row_block_idx * 2 + ctarank, col_block_idx * 8 + i});
                consumer::tma::store_async_read_wait();
                consumer::sync(1);
            }
        }
    }
}

PYBIND11_MODULE(_C, m) {
    m.doc() = "";
    kittens::py::bind_kernel<kernel>(m, "bf16_matmul",
        &globals::A,
        &globals::B,
        &globals::C
    );
}
