#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;

struct config {
    static constexpr int CLUSTER_SIZE = 1;
    static constexpr int SM_COUNT = 148;
    static constexpr int STATIC_SHARED_MEMORY = 128;
    static constexpr int DYNAMIC_SHARED_MEMORY = MAX_SHARED_MEMORY - STATIC_SHARED_MEMORY;

    static constexpr int CONSUMER_WARPGROUPS = 1;
    static constexpr int PRODUCER_WARPGROUPS = 1;
    static constexpr int NUM_WARPGROUPS = CONSUMER_WARPGROUPS + PRODUCER_WARPGROUPS;
    static constexpr int NUM_WARPS = NUM_WARPGROUPS * WARPGROUP_WARPS;
    static constexpr int NUM_THREADS = NUM_WARPS * WARP_THREADS;

    static constexpr int PRODUCER_REGISTERS = 56;
    static constexpr int CONSUMER_REGISTERS = 224;

    static constexpr int PIPELINE_STAGES = 4;
};

struct globals {
    static constexpr int SUPERGROUP_BLOCKS = 8;
    static constexpr int ROW_BLOCK = 64;
    static constexpr int COL_BLOCK = 128;
    static constexpr int REDUCTION_BLOCK = 128;

    using A_tile = st_bf<ROW_BLOCK, REDUCTION_BLOCK>;
    using B_tile = st_bf<COL_BLOCK, REDUCTION_BLOCK>;
    using C_tile = st_bf<ROW_BLOCK, COL_BLOCK / 8>;

    gl<bf16, 1, 1, -1, -1, A_tile> A;
    gl<bf16, 1, 1, -1, -1, B_tile> B;
    gl<bf16, 1, 1, -1, -1, C_tile> C;

    __host__ inline dim3 grid() { return dim3(config::SM_COUNT); }
    __host__ inline dim3 block() { return dim3(config::NUM_THREADS); }
    __host__ inline int dynamic_shared_memory() { return config::DYNAMIC_SHARED_MEMORY; }

    struct pipeline_inputs {
        A_tile A;
        B_tile B;
    };

    struct pipeline_outputs {
        C_tile C;
    };
};

__global__ __launch_bounds__(config::NUM_THREADS, 1)
void kernel(const __grid_constant__ globals G) {
    // Shared memory declaration
    extern __shared__ int __shm[];
    tma_swizzle_allocator allocator((int*)&__shm[0]);

    // Warpgroup configuration
    using consumer = group<config::CONSUMER_WARPGROUPS * WARPGROUP_WARPS>;
    int warpgroup_id = warpgroup::groupid();
    int warp_id = warpgroup::warpid();
    int lane_id = warp::laneid();

    // Allocate shared and tensor memory
    static_assert(sizeof(globals::pipeline_inputs) * config::PIPELINE_STAGES + sizeof(globals::pipeline_outputs) <= config::DYNAMIC_SHARED_MEMORY);
    globals::pipeline_inputs (&inputs)[config::PIPELINE_STAGES] = allocator.allocate<globals::pipeline_inputs, config::PIPELINE_STAGES>();
    globals::pipeline_outputs &outputs = allocator.allocate<globals::pipeline_outputs>();
    tensor_allocator<1, config::CLUSTER_SIZE> tm_allocator {};

    // Set up mbarriers
    __shared__ semaphore inputs_arrived[config::PIPELINE_STAGES];
    __shared__ semaphore inputs_finished[config::PIPELINE_STAGES];
    __shared__ semaphore outputs_arrived;
    __shared__ semaphore tensors_finished;
    if (threadIdx.x == 0) {
        for (int i = 0; i < config::PIPELINE_STAGES; ++i) {
            init_semaphore(inputs_arrived[i], 0, 1);
            init_semaphore(inputs_finished[i], 0, 1);
        }
        init_semaphore(outputs_arrived, 0, 1);
        init_semaphore(tensors_finished, 0, 1);
    }
    __syncthreads();

    // Pipeline configuration
    int num_blocks_per_row = G.C.cols() / globals::COL_BLOCK;
    int num_blocks_per_col = G.C.rows() / globals::ROW_BLOCK;
    int num_blocks = num_blocks_per_row * num_blocks_per_col;
    int num_iters_per_block = G.A.cols() / globals::REDUCTION_BLOCK;
    int num_blocks_per_supergroup = globals::SUPERGROUP_BLOCKS * num_blocks_per_row;

    // Declare stage and phasebits for semaphore waits
    int stage = 0;
    int last_stage = -1;
    uint32_t phasebits = 0xFFFF0000;

    // Main divergence
    if (warpgroup_id == config::NUM_WARPGROUPS - 1) {
        // Producer group
        warpgroup::decrease_registers<config::PRODUCER_REGISTERS>();

        // Sub divergence
        if (warp_id == 3 && lane_id == 0) {
            // Producer group -- loaders
            for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
                // Compute block indices
                int supergroup_idx = block_idx / num_blocks_per_supergroup;
                int idx_within_supergroup = block_idx % num_blocks_per_supergroup;
                int rows_in_supergroup = min(globals::SUPERGROUP_BLOCKS, num_blocks_per_col - supergroup_idx * globals::SUPERGROUP_BLOCKS);
                int row_within_supergroup = idx_within_supergroup % rows_in_supergroup;
                int row_block_idx = supergroup_idx * globals::SUPERGROUP_BLOCKS + row_within_supergroup;
                int col_block_idx = idx_within_supergroup / rows_in_supergroup;

                for (int i = 0; i < num_iters_per_block; ++i) {
                    wait(inputs_finished[stage], get_phasebit<1>(phasebits, stage));
                    update_phasebit<1>(phasebits, stage);
                    if (stage == last_stage) {
                        arrive(outputs_arrived);
                        last_stage = -1;
                    }
                    tma::expect_bytes(inputs_arrived[stage], sizeof(globals::pipeline_inputs));
                    tma::load_async(inputs[stage].A, G.A, {row_block_idx, i}, inputs_arrived[stage]);
                    tma::load_async(inputs[stage].B, G.B, {col_block_idx, i}, inputs_arrived[stage]);
                    if (i == num_iters_per_block - 1) {
                        last_stage = stage;
                    }
                    stage = (stage + 1) % config::PIPELINE_STAGES;
                }
            }
            if (last_stage >= 0) {
                wait(inputs_finished[last_stage], get_phasebit<1>(phasebits, last_stage));
                arrive(outputs_arrived);
            }
        } else if (warp_id == 0 && lane_id == 0) {
            // Producer group -- launchers
            auto tm = tm_allocator.allocate<tt<float, globals::ROW_BLOCK, globals::COL_BLOCK>>(0, 0);
            for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
                wait(tensors_finished, get_phasebit<1>(phasebits, config::PIPELINE_STAGES));
                update_phasebit<1>(phasebits, config::PIPELINE_STAGES);
                for (int i = 0; i < num_iters_per_block; ++i) {
                    wait(inputs_arrived[stage], get_phasebit<0>(phasebits, stage));
                    update_phasebit<0>(phasebits, stage);
                    if (i == 0)
                        mm_ABt(tm, inputs[stage].A, inputs[stage].B, inputs_finished[stage]);
                    else
                        mma_ABt(tm, inputs[stage].A, inputs[stage].B, inputs_finished[stage]);
                    stage = (stage + 1) % config::PIPELINE_STAGES;
                }
            }
        }
    } else {
        // Consumer group
        warpgroup::increase_registers<config::CONSUMER_REGISTERS>();
        auto tm = tm_allocator.allocate<tt<float, globals::ROW_BLOCK, globals::COL_BLOCK>>(0, 0);

        for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
            // Compute block indices
            int supergroup_idx = block_idx / num_blocks_per_supergroup;
            int idx_within_supergroup = block_idx % num_blocks_per_supergroup;
            int rows_in_supergroup = min(globals::SUPERGROUP_BLOCKS, num_blocks_per_col - supergroup_idx * globals::SUPERGROUP_BLOCKS);
            int row_within_supergroup = idx_within_supergroup % rows_in_supergroup;
            int row_block_idx = supergroup_idx * globals::SUPERGROUP_BLOCKS + row_within_supergroup;
            int col_block_idx = idx_within_supergroup / rows_in_supergroup;

            // Wait for the last matmul to complete
            wait(outputs_arrived, get_phasebit<0>(phasebits, config::PIPELINE_STAGES));
            update_phasebit<0>(phasebits, config::PIPELINE_STAGES);

            // Load the output from tensor memory into registers
            rt_bf<globals::ROW_BLOCK / 4, globals::COL_BLOCK / 8> C[8];
            #pragma unroll
            for (int i = 0; i < 8; i++)
                consumer::load_async(C[i], tm.subtile<tt<float, globals::ROW_BLOCK, globals::COL_BLOCK / 8>>(0, i * globals::COL_BLOCK / 8));
            tensor_load_wait();
            consumer::sync(1);
            if (consumer::laneid() == 0)
                arrive(tensors_finished, 0);

            // Store to global memory
            #pragma unroll
            for (int i = 0; i < 8; i++) {
                consumer::store(outputs.C, C[i]);
                consumer::sync(1);
                consumer::tma::store_async(G.C, outputs.C, {row_block_idx, col_block_idx * 8 + i});
                consumer::tma::store_async_read_wait();
                consumer::sync(1);
            }
        }
    }
}

PYBIND11_MODULE(_C, m) {
    m.doc() = "";
    kittens::py::bind_kernel<kernel>(m, "bf16_matmul",
        &globals::A,
        &globals::B,
        &globals::C
    );
}
