#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;

// ThunderKittens macro check
#if !defined(KITTENS_HOPPER) || !defined(KITTENS_BLACKWELL)
    #error "KITTENS_HOPPER and KITTENS_BLACKWELL macros must be defined for Blackwell compilation"
#endif

// Kernel configuration
struct config {
    static constexpr int SM_COUNT = 148;
    static constexpr int STATIC_SHARED_MEMORY = 1024;
    static constexpr int DYNAMIC_SHARED_MEMORY = MAX_SHARED_MEMORY - STATIC_SHARED_MEMORY;

    static constexpr int CONSUMER_WARPGROUPS = 2;
    static constexpr int PRODUCER_WARPGROUPS = 1;
    static constexpr int NUM_WARPGROUPS = CONSUMER_WARPGROUPS + PRODUCER_WARPGROUPS;
    static constexpr int NUM_WARPS = NUM_WARPGROUPS * WARPGROUP_WARPS;
    static constexpr int NUM_THREADS = NUM_WARPS * WARP_THREADS;

    static constexpr int PRODUCER_REGISTERS = 40;
    static constexpr int CONSUMER_REGISTERS = 232;

    static constexpr int PIPELINE_STAGES = 4;
};

// Kernel globals
struct globals {
    // 1. The block size should be equivalent to the quantization block size
    // 2. The block should be square (for transpose)
    static constexpr int BLOCK_SIZE = 128;
    static constexpr int QUANT_BLOCK_SIZE = 32;

    using A_tile_bf16 = st_bf<BLOCK_SIZE, BLOCK_SIZE>;
    using A_tile_fp8 = st_fp8e4m3<BLOCK_SIZE, BLOCK_SIZE>;
    using A_sc_vec = sv<fp8e8m0, BLOCK_SIZE * BLOCK_SIZE / QUANT_BLOCK_SIZE>;

    gl<bf16, 1, -1, -1, -1, A_tile_bf16> A_bf16;
    gl<fp8e4m3, 1, -1, -1, -1, A_tile_fp8> A_fp8;
    gl<float, -1, -1, -1, -1, A_sc_vec> A_sc;

    __host__ inline dim3 grid() { return dim3(config::SM_COUNT); }
    __host__ inline dim3 block() { return dim3(config::NUM_THREADS); }
    __host__ inline int dynamic_shared_memory() { return config::DYNAMIC_SHARED_MEMORY; }

    struct pipeline_inputs {
        A_tile_bf16 A_bf16;
    };

    struct pipeline_outputs {
        A_tile_fp8 A_fp8;
        A_sc_vec A_sc;
    };
};

// Kernel implementation
__global__  __launch_bounds__(config::NUM_THREADS, 1)
void kernel(const __grid_constant__ globals G) {
    // Shared memory declaration
    extern __shared__ int __shm[];
    tma_swizzle_allocator allocator((int*)&__shm[0]);

    // Warpgroup configuration
    int lane_id = warp::laneid();
    int warp_id = warpgroup::warpid();
    int warpgroup_id = warpgroup::groupid();
    using consumer = group<config::CONSUMER_WARPGROUPS * WARPGROUP_WARPS>;

    // Allocate shared memory
    static_assert(sizeof(globals::pipeline_inputs) * config::PIPELINE_STAGES + sizeof(globals::pipeline_outputs) * config::PIPELINE_STAGES <= config::DYNAMIC_SHARED_MEMORY);
    globals::pipeline_inputs (&inputs)[config::PIPELINE_STAGES] = allocator.allocate<globals::pipeline_inputs, config::PIPELINE_STAGES>();
    globals::pipeline_outputs (&outputs)[config::PIPELINE_STAGES] = allocator.allocate<globals::pipeline_outputs, config::PIPELINE_STAGES>();

    // Set up mbarriers
    __shared__ semaphore inputs_arrived[config::PIPELINE_STAGES];
    __shared__ semaphore inputs_finished[config::PIPELINE_STAGES];
    if (threadIdx.x == 0) {
        for (int i = 0; i < config::PIPELINE_STAGES; ++i) {
            init_semaphore(inputs_arrived[i], 0, 1);
            init_semaphore(inputs_finished[i], 0, 2);
        }
    }
    __syncthreads();

    // Pipeline configuration
    int num_groups = G.A_bf16.depth();
    int num_blocks_per_row = G.A_bf16.cols() / globals::BLOCK_SIZE;
    int num_blocks_per_col = G.A_bf16.rows() / globals::BLOCK_SIZE;
    int num_blocks_per_group = num_blocks_per_row * num_blocks_per_col;
    int num_blocks = num_groups * num_blocks_per_group;

    // Declare phasebits for semaphore waits
    uint32_t phasebits = 0xFFFF0000;

    // Main divergence
    if (warpgroup_id == config::NUM_WARPGROUPS - 1) {
        // Producer group
        warpgroup::decrease_registers<config::PRODUCER_REGISTERS>();

        // Main loop
        int stage = 0;
        if (warp_id == 0 && lane_id == 0) {
            for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
                // Compute block indices
                int group_idx = block_idx / num_blocks_per_group;
                int group_local_block_idx = block_idx % num_blocks_per_group;
                int row_block_idx = group_local_block_idx / num_blocks_per_row;
                int col_block_idx = group_local_block_idx % num_blocks_per_row;

                // Wait for shared memory to be free
                wait(inputs_finished[stage], get_phasebit<1>(phasebits, stage));
                update_phasebit<1>(phasebits, stage);

                // Load inputs into shared memory
                tma::expect_bytes(inputs_arrived[stage], sizeof(globals::pipeline_inputs));
                tma::load_async(inputs[stage].A_bf16, G.A_bf16, {group_idx, row_block_idx, col_block_idx}, inputs_arrived[stage]);

                // Update phasebit and stage
                stage = (stage + 1) % config::PIPELINE_STAGES;
            }
        }
    } else {
        // Consumer group
        warpgroup::increase_registers<config::CONSUMER_REGISTERS>();

        // Main loop
        int stage = 0;
        for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
            // Compute block indices
            int group_idx = block_idx / num_blocks_per_group;
            int group_local_block_idx = block_idx % num_blocks_per_group;
            int row_block_idx = group_local_block_idx / num_blocks_per_row;
            int col_block_idx = group_local_block_idx % num_blocks_per_row;

            // Wait for inputs to arrive at shared memory
            wait(inputs_arrived[stage], get_phasebit<0>(phasebits, stage));
            update_phasebit<0>(phasebits, stage);

            // Load input
            rt_bf<globals::BLOCK_SIZE / 8, globals::BLOCK_SIZE> A_bf16;
            consumer::load(A_bf16, inputs[stage].A_bf16[warpgroup_id]);
            consumer::sync(1);
            consumer::arrive(inputs_finished[stage]);

            // Quantize
            // I think I can do this by reinterpret-casting tiles into 4 array
            rt_fl<globals::BLOCK_SIZE / 8, globals::BLOCK_SIZE> A_fl, A_fl_abs;
            rt_fp8e4m3<globals::BLOCK_SIZE / 8, globals::BLOCK_SIZE> A_fp8;
            col_vec<rt_fl<globals::BLOCK_SIZE / 8, globals::BLOCK_SIZE>> scale;
            warp::copy(A_fl, A_bf16);
            warp::abs(A_fl_abs, A_fl);
            warp::row_max(scale, A_fl_abs);
            warp::max(scale, scale, 0.000000000001f); // avoid division by zero
            warp::mul(scale, scale, 0.002232142857f); // 1 / 448
            warp::div_row(A_fl, A_fl, scale);
            warp::copy(A_fp8, A_fl);

            // Store results to shared memory
            consumer::store(outputs[stage].A_fp8[warpgroup_id], A_fp8);
            consumer::store(outputs[stage].A_sc[warpgroup_id], scale);
            consumer::sync(1);

            // Store results to global memory
            // TODO: why the fuck do I have C memory per stage? we only use single stage at a time. Reduce this and increase pipeline depth
            if (consumer::laneid() == 0) {
                tma::store_async(G.A_fp8, outputs[stage].A_fp8[warpgroup_id], {group_idx, row_block_idx, col_block_idx});
                tma::store_async(G.A_sc, outputs[stage].A_sc[warpgroup_id], {group_idx, col_block_idx, row_block_idx}); // column-major
                tma::store_async_read_wait();
            }
            consumer::sync(1);

            // Update phasebit and stage
            stage = (stage + 1) % config::PIPELINE_STAGES;
        }
    }
}

// Python bindings
PYBIND11_MODULE(_C, m) {
    m.doc() = "";
    kittens::py::bind_kernel<kernel>(m, "kernel",
        &globals::A_bf16,
        &globals::A_fp8,
        &globals::A_sc
    );
}
