#include "hip/hip_runtime.h"
/*
    Curious: would relying on occupancy have similar speed as TMA?

    TMA: 6651.22 GB/s
    Warp asynchrony: 1983.76 GB/s

    Well this was obvious since this experiment does not have to go through the registers at all
*/
#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/torchutils.cuh"

using namespace kittens;
using namespace kittens::prototype;

// 0: TMA
// 1: Warp
static constexpr int MODE = 1;

struct config {
    static constexpr int CLUSTER_SIZE = 1;
    static constexpr int NUM_WARPGROUPS = 1;
    static constexpr int NUM_WARPS = NUM_WARPGROUPS * WARPGROUP_WARPS; 
    static constexpr int NUM_THREADS = NUM_WARPS * WARP_THREADS;
};

struct globals {
    static constexpr int BLOCK_SIZE = 128;

    using tile = st_bf<BLOCK_SIZE, BLOCK_SIZE>;
    using layout = gl<bf16,  1, 1, -1, -1, tile>;

    layout src;
    layout dst;

    __host__ inline dim3 grid() const {
        if constexpr (MODE == 0) {
            return dim3(src.cols() / BLOCK_SIZE, src.rows() / BLOCK_SIZE);
        } else {
            return dim3(src.cols() * src.rows() / (config::NUM_THREADS * 2));
        }
    }
    __host__ inline int dynamic_shared_memory() const {
        if constexpr (MODE == 0) {
            return sizeof(tile) + 1024;
        } else {
            return 0;
        }
    }
};

__device__ inline void kernel(const globals &G) {
    if constexpr (MODE == 0) {
        // Declare shared memory
        extern __shared__ int __shm[]; 
        tma_swizzle_allocator sm_allocator((int*)&__shm[0]);
        globals::tile &tile = sm_allocator.allocate<globals::tile>();

        const int row = blockIdx.y;
        const int col = blockIdx.x;

        __shared__ semaphore inputs_arrived;
        if (threadIdx.x == 0) {
            init_semaphore(inputs_arrived, 0, 1);
            tma::expect_bytes(inputs_arrived, sizeof(globals::tile));
            tma::load_async(tile, G.src, {row, col}, inputs_arrived);
        }
        __syncthreads();

        wait(inputs_arrived, 0);

        if (threadIdx.x == 0) {
            tma::store_async(G.dst, tile, {row, col});
        }
    } else {
        const int idx = 2 * config::NUM_THREADS * blockIdx.x + 2 * threadIdx.x;
        bf16_2 tmp;
        asm volatile("{ld.weak.global.b32 %0, [%1];}"
            : "=r"(*reinterpret_cast<uint32_t*>(&tmp))
            : "l"(&G.src.raw_ptr[idx])
            : "memory"
        );
        asm volatile("{st.weak.global.b32 [%0], %1;}"
            :
            : "l"(&G.dst.raw_ptr[idx]), "r"(*reinterpret_cast<uint32_t*>(&tmp))
            : "memory"
        );
    }
}

void entrypoint(at::Tensor &src, at::Tensor &dst) {
    globals G {
        .src = kittens::py::tensor_to_gl<globals::layout>(src),
        .dst = kittens::py::tensor_to_gl<globals::layout>(dst)
    };

    kittens::py::launch_kernel<config, globals, kernel>(G);
}

#include <torch/csrc/utils/pybind.h>

PYBIND11_MODULE(_C, m) {
    m.def("kernel", &entrypoint);
}
