#include "hip/hip_runtime.h"
/*
    As stated in the previous test:
    Staying within 1-CTA matmuls, we get:
    - M=128 K=128 N=128 : 2130 TFLOPs
    - M=128 K=128 N=256 : 2600 TFLOPs

    Now, we can't go to M=256 N=256, because we run out of TM space (128x512) to store the scales
    Thus, another alternative is to set N=128 and increase the M.

    This actually makes it easier to cope with non-M%256==0 inputs, because we can just
    "turn off" subset of the tensor core for the tail iterations.

    Also, we can extend and run M=384 N=128 matmuls, using 3/4 of TM for matmuls and 1/4 for scales.

    Observation:
        - It's FASTER to increase M than to increase N
        - Pure M=256 K=128 N=128 : 2640 TFLOPs
*/

#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;

// Kernel configuration
struct config {
    static constexpr int SM_COUNT = 148;
    static constexpr int STATIC_SHARED_MEMORY = 1024;
    static constexpr int DYNAMIC_SHARED_MEMORY = MAX_SHARED_MEMORY - STATIC_SHARED_MEMORY;

    static constexpr int CONSUMER_WARPGROUPS = 2;
    static constexpr int PRODUCER_WARPGROUPS = 1;
    static constexpr int NUM_WARPGROUPS = CONSUMER_WARPGROUPS + PRODUCER_WARPGROUPS;
    static constexpr int NUM_WARPS = NUM_WARPGROUPS * WARPGROUP_WARPS;
    static constexpr int NUM_THREADS = NUM_WARPS * WARP_THREADS;

    static constexpr int PRODUCER_REGISTERS = 40;
    static constexpr int CONSUMER_REGISTERS = 232;

    static constexpr int PIPELINE_STAGES = 4;
};

// Kernel globals
struct globals {
    static constexpr int SUPERGROUP_BLOCKS = 12;
    static constexpr int ROW_BLOCK = 256;
    static constexpr int COL_BLOCK = 128;
    static constexpr int REDUCTION_BLOCK = 128;

    using A_tile = st_fp8e4m3<ROW_BLOCK / 2, REDUCTION_BLOCK>;
    using B_tile = st_fp8e4m3<COL_BLOCK, REDUCTION_BLOCK>;
    using C_tile = st_bf<ROW_BLOCK / 8, COL_BLOCK>;

    gl<fp8e4m3, 1, 1, -1, -1, A_tile> A;
    gl<fp8e4m3, 1, 1, -1, -1, B_tile> B;
    gl<bf16, 1, 1, -1, -1, C_tile> C;

    __host__ inline dim3 grid() { return dim3(config::SM_COUNT); }
    __host__ inline dim3 block() { return dim3(config::NUM_THREADS); }
    __host__ inline int dynamic_shared_memory() { return config::DYNAMIC_SHARED_MEMORY; }

    struct pipeline_inputs {
        A_tile A[2];
        B_tile B;
    };

    struct pipeline_outputs {
        C_tile C;
    };
};

// Kernel implementation
__global__ __launch_bounds__(config::NUM_THREADS, 1)
void kernel(const __grid_constant__ globals G) {
    // Shared memory declaration
    extern __shared__ int __shm[];
    tma_swizzle_allocator allocator((int*)&__shm[0]);

    // Warpgroup configuration
    using consumer = group<config::CONSUMER_WARPGROUPS * WARPGROUP_WARPS>;
    int warpgroup_id = warpgroup::groupid();
    int warp_id = warpgroup::warpid();
    int lane_id = warp::laneid();

    // Allocate shared and tensor memory
    static_assert(sizeof(globals::pipeline_inputs) * config::PIPELINE_STAGES + sizeof(globals::pipeline_outputs) <= config::DYNAMIC_SHARED_MEMORY);
    globals::pipeline_inputs (&inputs)[config::PIPELINE_STAGES] = allocator.allocate<globals::pipeline_inputs, config::PIPELINE_STAGES>();
    globals::pipeline_outputs &outputs = allocator.allocate<globals::pipeline_outputs>();
    tensor_allocator<1, 1> tm_allocator {};

    // Set up mbarriers
    __shared__ semaphore inputs_arrived[config::PIPELINE_STAGES];
    __shared__ semaphore inputs_finished[config::PIPELINE_STAGES];
    __shared__ semaphore outputs_arrived;
    __shared__ semaphore tensors_finished;
    if (threadIdx.x == 0) {
        for (int i = 0; i < config::PIPELINE_STAGES; ++i) {
            init_semaphore(inputs_arrived[i], 0, 1);
            init_semaphore(inputs_finished[i], 0, 2);
        }
        init_semaphore(outputs_arrived, 0, 1);
        init_semaphore(tensors_finished, 0, 1);
    }
    __syncthreads();

    // Pipeline configuration
    int num_blocks_per_row = G.C.cols() / globals::COL_BLOCK;
    int num_blocks_per_col = G.C.rows() / globals::ROW_BLOCK;
    int num_blocks = num_blocks_per_row * num_blocks_per_col;
    int num_iters_per_block = G.A.cols() / globals::REDUCTION_BLOCK;
    int num_blocks_per_supergroup = globals::SUPERGROUP_BLOCKS * num_blocks_per_row;

    // Declare stage and phasebits for semaphore waits
    int stage = 0;
    int last_stage = -1;
    uint32_t phasebits = 0xFFFF0000;

    // Main divergence
    if (warpgroup_id == config::NUM_WARPGROUPS - 1) {
        // Producer group
        warpgroup::decrease_registers<config::PRODUCER_REGISTERS>();

        // Sub divergence
        if (warp_id == 3 && lane_id == 0) {
            // Producer group -- loaders
            for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
                // Compute block indices
                int supergroup_idx = block_idx / num_blocks_per_supergroup;
                int idx_within_supergroup = block_idx % num_blocks_per_supergroup;
                int rows_in_supergroup = min(globals::SUPERGROUP_BLOCKS, num_blocks_per_col - supergroup_idx * globals::SUPERGROUP_BLOCKS);
                int row_within_supergroup = idx_within_supergroup % rows_in_supergroup;
                int row_block_idx = supergroup_idx * globals::SUPERGROUP_BLOCKS + row_within_supergroup;
                int col_block_idx = idx_within_supergroup / rows_in_supergroup;

                for (int i = 0; i < num_iters_per_block; ++i) {
                    wait(inputs_finished[stage], get_phasebit<1>(phasebits, stage));
                    if (stage == last_stage) {
                        arrive(outputs_arrived);
                        last_stage = -1;
                    }
                    tma::expect_bytes(inputs_arrived[stage], sizeof(globals::pipeline_inputs));
                    tma::load_async(inputs[stage].A[0], G.A, {row_block_idx * 2 + 0, i}, inputs_arrived[stage]);
                    tma::load_async(inputs[stage].A[1], G.A, {row_block_idx * 2 + 1, i}, inputs_arrived[stage]);
                    tma::load_async(inputs[stage].B, G.B, {col_block_idx, i}, inputs_arrived[stage]);
                    update_phasebit<1>(phasebits, stage);
                    if (i == num_iters_per_block - 1) {
                        last_stage = stage;
                    }
                    stage = (stage + 1) % config::PIPELINE_STAGES;
                }
            }
            wait(inputs_finished[last_stage], get_phasebit<1>(phasebits, last_stage));
            arrive(outputs_arrived);
        } else if (warp_id == 0 && lane_id < 2) {
            // Producer group -- launchers
            using tm_t = tt<float, globals::ROW_BLOCK / 2, globals::COL_BLOCK>;
            tm_t tm = tm_allocator.template allocate<tm_t>(128 * lane_id);
            for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
                wait(tensors_finished, get_phasebit<1>(phasebits, config::PIPELINE_STAGES));
                update_phasebit<1>(phasebits, config::PIPELINE_STAGES);
                wait(inputs_arrived[stage], get_phasebit<0>(phasebits, stage));
                update_phasebit<0>(phasebits, stage);
                mm_ABt(tm, inputs[stage].A[lane_id], inputs[stage].B, inputs_finished[stage]);
                stage = (stage + 1) % config::PIPELINE_STAGES;
                for (int i = 1; i < num_iters_per_block; ++i) {
                    wait(inputs_arrived[stage], get_phasebit<0>(phasebits, stage)); 
                    update_phasebit<0>(phasebits, stage);
                    mma_ABt(tm, inputs[stage].A[lane_id], inputs[stage].B, inputs_finished[stage]);
                    stage = (stage + 1) % config::PIPELINE_STAGES;
                }
            }
        }
    } else if (warpgroup_id < config::CONSUMER_WARPGROUPS) {
        // Consumer group
        warpgroup::increase_registers<config::CONSUMER_REGISTERS>();
        using tm_t = tt<float, globals::ROW_BLOCK / 2, globals::COL_BLOCK>;
        tm_t tm = tm_allocator.template allocate<tm_t>(warpgroup_id * 128);

        for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
            // Compute block indices
            int supergroup_idx = block_idx / num_blocks_per_supergroup;
            int idx_within_supergroup = block_idx % num_blocks_per_supergroup;
            int rows_in_supergroup = min(globals::SUPERGROUP_BLOCKS, num_blocks_per_col - supergroup_idx * globals::SUPERGROUP_BLOCKS);
            int row_within_supergroup = idx_within_supergroup % rows_in_supergroup;
            int row_block_idx = supergroup_idx * globals::SUPERGROUP_BLOCKS + row_within_supergroup;
            int col_block_idx = idx_within_supergroup / rows_in_supergroup;

            // Wait for the last matmul to complete
            wait(outputs_arrived, get_phasebit<0>(phasebits, config::PIPELINE_STAGES));
            update_phasebit<0>(phasebits, config::PIPELINE_STAGES);

            // Load the output from tensor memory into registers
            rt_fl<globals::ROW_BLOCK / 8, globals::COL_BLOCK> C_reg;
            warpgroup::load_async(C_reg, tm);
            tensor_load_wait();
            consumer::sync(1); // wait for both consumer WGs, to arrive asap
            if (consumer::laneid() == 0)
                arrive(tensors_finished);

            // Store to global memory
            for (int i = 0; i < 8; ++i) {
                if (consumer::warpid() == i) {
                    warp::store(outputs.C, C_reg);
                    __syncwarp();
                    if (lane_id == 0) {
                        tma::store_async(G.C, outputs.C, {row_block_idx * 8 + i, col_block_idx});
                        tma::store_async_read_wait();
                    }
                }
                consumer::sync(1);
            }
        }
    }
}

// Python bindings
PYBIND11_MODULE(_C, m) {
    m.doc() = "";
    kittens::py::bind_kernel<kernel>(m, "kernel",
        &globals::A,
        &globals::B,
        &globals::C
    );
}
