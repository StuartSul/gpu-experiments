#include "hip/hip_runtime.h"
/*
    WIP. Does not work yet
*/

#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <mpi.h>

#define CUDACHECK(cmd) do {                                   \
    hipError_t err = cmd;                                    \
    if (err != hipSuccess) {                                 \
        fprintf(stderr, "Failed: CUDA error %s:%d '%s'\n",    \
            __FILE__, __LINE__, hipGetErrorString(err));     \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
} while(0)

// perform Allreduce using ring
__global__ void ring_reduce(
    int *dst,
    const int *src,
    size_t N,
    uint64_t *signal
) {
    int rank = nvshmem_my_pe();
    int world_size = nvshmem_n_pes();
    int peer_rank = (rank + 1) % world_size;

    size_t N_per_dev = N / world_size;
    size_t N_per_block = N_per_dev / gridDim.x;
    size_t base_idx = rank * N_per_dev + blockIdx.x * N_per_block;

    if (base_idx > N)
        return;

    src = src + base_idx;
    dst = dst + base_idx;
    signal = signal + blockIdx.x;

    // Reduce phase
    for (int step = 0; step < world_size - 1; step++) {
        if (rank != 0) {
            if (threadIdx.x == 0)
                nvshmem_signal_wait_until(signal, NVSHMEM_CMP_GE, step + 1);
            __syncthreads();

            for (size_t i = threadIdx.x; i < N_per_dev; i += blockDim.x) {
                dst[i] = dst[i] + src[i];
            }
            __syncthreads();
        }
        if (threadIdx.x == 0)
            nvshmem_int_put_signal_nbi(dst, (rank == 0) ? src : dst, N_per_dev, signal, 1, NVSHMEM_SIGNAL_ADD, peer_rank);

        src = src + N_per_dev;
        dst = dst + N_per_dev;
    }

    // Broadcast phase
    dst = dst - num_chunks * chunk_elems;
    if (threadIdx.x == 0) {
        for (size_t chunk = 0; chunk < num_chunks; chunk++) {
            if (rank < world_size - 1)
                nvshmem_signal_wait_until(signal, NVSHMEM_CMP_GE, (rank == 0) ? chunk + 1 : num_chunks + chunk + 1);
            if (rank < world_size - 2)
                nvshmem_int_put_signal_nbi(dst, dst, chunk_elems, signal, 1, NVSHMEM_SIGNAL_ADD, peer);
            dst = dst + chunk_elems;
        }
        *signal = 0;
    }
}

static size_t SIZE = 1024 * 1024 * 32;
static constexpr int NUM_BLOCKS = 32;
static constexpr int NUM_THREADS = 512;
static constexpr int NUM_WARMUPS = 1;
static constexpr int NUM_ITERS = 4;

int main(int argc, char **argv) {
    // Initialize MPI
    int rank;
    int world_size;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    // Initialize NVSHMEM
    nvshmemx_init_attr_t attr = NVSHMEMX_INIT_ATTR_INITIALIZER;
    MPI_Comm mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    // Retrieve NVSHMEM PE info
    int current_pe = nvshmem_my_pe();
    int num_pes = nvshmem_n_pes();
    int current_pe_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    assert(current_pe_node == current_pe);
    assert(current_pe == rank);
    assert(num_pes == world_size);

    // Set CUDA device
    CUDACHECK(hipSetDevice(current_pe_node));

    // Create CUDA stream and events
    hipStream_t stream;
    hipEvent_t start_event;
    hipEvent_t stop_event;
    CUDACHECK(hipEventCreate(&start_event));
    CUDACHECK(hipEventCreate(&stop_event));
    CUDACHECK(hipStreamCreate(&stream));

    // Allocate and initialize host memory
    size_t N = SIZE / sizeof(int);
    int *host_buffer = reinterpret_cast<int *>(malloc(SIZE));
    for (size_t i = 0; i < N; i++)
        host_buffer[i] = i;

    // Allocate and initialize device memory
    int *dst = reinterpret_cast<int *>(nvshmem_malloc(SIZE));
    int *src = reinterpret_cast<int *>(nvshmem_malloc(SIZE));
    uint64_t *signal = (uint64_t *)nvshmem_calloc(NUM_BLOCKS, sizeof(uint64_t));
    CUDACHECK(hipMemcpyAsync(src, host_buffer, SIZE, hipMemcpyHostToDevice, stream));
    nvshmemx_barrier_all_on_stream(stream);

    // Kernel configuration
    dim3 gridDim(NUM_BLOCKS);
    dim3 blockDim(NUM_THREADS);
    void *args[] = {&dst, &src, &N, &signal};

    // Check correctness
    nvshmemx_collective_launch((const void *)ring_reduce, gridDim, blockDim, args, 0, stream);
    nvshmemx_barrier_all_on_stream(stream);
    CUDACHECK(hipMemcpyAsync(host_buffer, dst, SIZE, hipMemcpyDeviceToHost, stream));
    CUDACHECK(hipStreamSynchronize(stream));
    for (int i = 0; i < N; i++) {
        if (host_buffer[i] != i * world_size)
            printf("Error on rank %d: data[%d] = %d (expected %d)\n", rank, i, host_buffer[i], i * world_size);
    }

    // Warmups
    for (int i = 0; i < NUM_WARMUPS; i++) {
        nvshmemx_collective_launch((const void *)ring_reduce, gridDim, blockDim, args, 0, stream);
        nvshmemx_barrier_all_on_stream(stream);
    }
    CUDACHECK(hipStreamSynchronize(stream));

    // Benchmark
    CUDACHECK(hipEventRecord(start_event, stream));
    for (int i = 0; i < NUM_ITERS; i++) {
        nvshmemx_collective_launch((const void *)ring_reduce, gridDim, blockDim, args, 0, stream);
        nvshmemx_barrier_all_on_stream(stream);
    }
    CUDACHECK(hipEventRecord(stop_event, stream));
    CUDACHECK(hipStreamSynchronize(stream));

    // Print result
    if (rank == 0) {
        float total_ms;
        CUDACHECK(hipEventElapsedTime(&total_ms, start_event, stop_event));
        printf("%zuB \t %fms\n", SIZE, total_ms / NUM_ITERS);
    }

    // Clean up
    CUDACHECK(hipEventDestroy(start_event));
    CUDACHECK(hipEventDestroy(stop_event));
    nvshmem_free(dst);
    nvshmem_free(src);
    nvshmem_free(signal);
    free(host_buffer);

    nvshmem_finalize();
    MPI_Finalize();
    return 0;
}
