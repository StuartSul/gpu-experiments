#include "hip/hip_runtime.h"
#include "gpu-experiments.cuh"

/*

Generic notes on tcgen05 instructions

- Tensor memory (TM) is on-chip memory (likely SRAM)
- It is organized as 2D matrix (rows are called "lanes" and columns are called as is)
    - On sm_100a, this is 128 x 512 per CTA, each cell 32-bit in size
- TM address is 32-bit, where first 16 significant bits are lane index and next 16 are column index
- TM must be allocated by a single warp in a CTA
    - Allocation is done in columns only (all lanes in the columns are allocated)
    - Granularity is (1) powers of 2 and (2) at least 32
- Supported matrix multiply and accumulate shapes: https://docs.nvidia.com/cuda/parallel-thread-execution/#tcgen05-matrix-shape
    - For 1-CTA dense (MX)FP8 matrix multiply, K is always 32
    - M and N are specified in the instruction descriptor
- Data movement shapes are in format lane x bits: https://docs.nvidia.com/cuda/parallel-thread-execution/#tcgen05-data-movement-shape
    - Each movement type (16x32b, 32x32b, ...) has its unique way of how values across registers are spread throughout TM
- A warp in a warpgroup can access only 1/4 of the lanes, and all columns of the TM
    - Warp 0: lanes 0-31
    - Warp 1: lanes 32-63
    - Warp 2: lanes 64-95
    - Warp 3: lanes 96-127

*/

// Global dimension
constexpr int N = 128;
constexpr int M = 128;
constexpr int K = 32;

// Tile dimension
constexpr int TILE_N = 128;
constexpr int TILE_M = 128;
constexpr int TILE_K = 32;

// Quantization
constexpr int Q_BLOCK = 32;
constexpr int NUM_BLOCKS = K / Q_BLOCK;
constexpr float DEST_MAX = 448.0;

// Kernel
constexpr int SM_COUNT = 148;
constexpr int WARP_THREADS = 32;
constexpr int WARPGROUP_WARPS = 4;
constexpr int WARPGROUP_THREADS = WARP_THREADS * WARPGROUP_WARPS;
constexpr int NUM_WARPGROUPS = 2;
constexpr int NUM_THREADS = WARPGROUP_THREADS * NUM_WARPGROUPS;
constexpr int MAX_SHARED_MEMORY = 227000; // Hopper/Blackwell
constexpr int DYNAMIC_SHARED_MEMORY = MAX_SHARED_MEMORY - 1000;


__global__ void kernel(
    const __grid_constant__ __hip_fp8_e4m3_fnuz * const A_fp8,
    const __grid_constant__ __nv_fp8_e8m0 * const A_sc,
    const __grid_constant__ CUtensorMap A_tmap,
    const __grid_constant__ __hip_fp8_e4m3_fnuz * const B_fp8,
    const __grid_constant__ __nv_fp8_e8m0 * const B_sc,
    const __grid_constant__ CUtensorMap B_tmap,
    const __grid_constant__ float * const C
) { 
    // Retrieve thread info
    int lane_id = threadIdx.x % WARP_THREADS;
    int warp_id = threadIdx.x / WARP_THREADS;
    int warpgroup_id = threadIdx.x / WARPGROUP_THREADS;

    // Allocate shared memory
    extern __shared__ int __shm[];
    __shared__ uint64_t mbarrier;
    __shared__ uint32_t tm_addr_shared;

    // Assign shared tiles. TMA swizzle require 1024 alignment max
    uint64_t __shm_ptr = reinterpret_cast<uint64_t>(&__shm[0]);
    __hip_fp8_e4m3_fnuz *A_fp8_shm = reinterpret_cast<__hip_fp8_e4m3_fnuz *>(((__shm_ptr + 1023) / 1024) * 1024);
    __shm_ptr += sizeof(__hip_fp8_e4m3_fnuz) * TILE_M * TILE_K;
    __nv_fp8_e8m0 *A_sc_shm = reinterpret_cast<__nv_fp8_e8m0 *>(((__shm_ptr + 1023) / 1024) * 1024);
    __shm_ptr += sizeof(__nv_fp8_e8m0) * TILE_M * NUM_BLOCKS;
    __hip_fp8_e4m3_fnuz *B_fp8_shm = reinterpret_cast<__hip_fp8_e4m3_fnuz *>(((__shm_ptr + 1023) / 1024) * 1024);
    __shm_ptr += sizeof(__hip_fp8_e4m3_fnuz) * TILE_N * TILE_K;
    __nv_fp8_e8m0 *B_sc_shm = reinterpret_cast<__nv_fp8_e8m0 *>(((__shm_ptr + 1023) / 1024) * 1024);
    __shm_ptr += sizeof(__nv_fp8_e8m0) * TILE_N * NUM_BLOCKS;
    float *C_shm = reinterpret_cast<float *>(((__shm_ptr + 1023) / 1024) * 1024);
    __shm_ptr += sizeof(float) * TILE_M * TILE_N;
    if (__shm_ptr >= DYNAMIC_SHARED_MEMORY) {
        if (threadIdx.x == 0) printf("ERROR: Exceeded maximum dynamic shared memory.");
        asm volatile("trap;");
    }

    // Initialize mbarriers
    if (threadIdx.x == 0) {
        asm volatile("mbarrier.init.shared::cta.b64 [%0], %1;"
            :
            : "l"(__cvta_generic_to_shared(&mbarrier)), "r"(1)
        );
    }
    __syncthreads();

    // Allocate Tensor Memory (TM) for 1-CTA group 
    uint32_t tm_addr = 0;
    uint32_t n_cols = 32; // must be unsigned 32b
    if (warp_id == 0) { // must be performed by a single warp in the CTA
        asm volatile("tcgen05.alloc.cta_group::1.sync.aligned.b32 [%0], %1;"
            :
            : "l"((uint64_t)&tm_addr_shared), "r"(n_cols)
        ); // __syncwarp() naturally happens here
        // After relinquish_alloc_permit, it becomes illegal for this CTA to call tcgen05.alloc
        asm volatile("tcgen05.relinquish_alloc_permit.cta_group::1.sync.aligned;");
    }
    __syncthreads();
    tm_addr = tm_addr_shared; // Move from shared memory into register

    // Main work begins here
    if (warpgroup_id == 1) {
        // Producer warpgroup
    } else {
        // Consumer warpgroup
    }

    // De-allocate TM for 1-CTA group
    if (warp_id == 0) { // must be performed by a single warp in the CTA
        asm volatile("tcgen05.dealloc.cta_group::1.sync.aligned.b32 %0, %1;"
            :
            : "r"(tm_addr), "r"(n_cols)
        );
    }
}


int main() {
    static_assert(K % Q_BLOCK == 0, "K must be divisible by Q_BLOCK");
    std::cout << "M = " << M << ", N = " << N << ", K = " << K << ", Q_BLOCK = " << Q_BLOCK << std::endl;

    // Allocate host memory
    float *h_A = new float[M * K];
    float *h_B = new float[N * K];
    __hip_fp8_e4m3_fnuz *h_A_fp8 = new __hip_fp8_e4m3_fnuz[M * K];
    __hip_fp8_e4m3_fnuz *h_B_fp8 = new __hip_fp8_e4m3_fnuz[N * K];
    __nv_fp8_e8m0 *h_A_sc = new __nv_fp8_e8m0[M * NUM_BLOCKS];
    __nv_fp8_e8m0 *h_B_sc = new __nv_fp8_e8m0[N * NUM_BLOCKS];
    float *h_C = new float[M * N];
    float *h_C_ref = new float[M * N];
    std::cout << "Allocated host memory" << std::endl;

    // Initialize matrices with random values
    std::random_device rd;
    std::mt19937 gen(42);
    std::normal_distribution<float> dis(0.0, 1.0);
    for (int i = 0; i < M * K; ++i) h_A[i] = dis(gen);
    for (int i = 0; i < N * K; ++i) h_B[i] = dis(gen);
    std::cout << "Initialized matrices" << std::endl;

    // Matrix A quantization
    for (int i = 0; i < M; i++) {
        for (int block = 0; block < NUM_BLOCKS; block++) {
            // Get block absolute maximum
            float amax = fabsf(h_A[i * K + block * Q_BLOCK]);
            for (int j = 1; j < Q_BLOCK; j++)
                amax = fmaxf(amax, h_A[i * K + block * Q_BLOCK + j]);

            // ceilf(log2f(amax / DEST_MAX)) with round to +inf & clamp to [2^-127, 2^127]
            h_A_sc[i * NUM_BLOCKS + block] = __nv_fp8_e8m0(amax / DEST_MAX); 
            // printf("actual: %f, stored: %d, val: %f\n", amax / DEST_MAX, 
            //     *reinterpret_cast<uint8_t *>(&h_A_sc[i * NUM_BLOCKS + block]) - 127,
            //     powf(2., *reinterpret_cast<uint8_t *>(&h_A_sc[i * NUM_BLOCKS + block]) - 127));

            // Quantize
            for (int j = 0; j < Q_BLOCK; j++) {
                float quantized_fp32 = h_A[i * K + block * Q_BLOCK + j] / 
                    powf(2., *reinterpret_cast<uint8_t *>(&h_A_sc[i * NUM_BLOCKS + block]) - 127);
                h_A_fp8[i * K + block * Q_BLOCK + j] = __hip_fp8_e4m3_fnuz(quantized_fp32);
            }
        }
    }

    // Matrix B quantization
    for (int i = 0; i < N; i++) {
        for (int block = 0; block < NUM_BLOCKS; block++) {
            // Get block absolute maximum
            float amax = fabsf(h_B[i * K + block * Q_BLOCK]);
            for (int j = 1; j < Q_BLOCK; j++)
                amax = fmaxf(amax, h_B[i * K + block * Q_BLOCK + j]);

            // this does ceilf(log2f(amax / DEST_MAX)) with round to +inf & clamp to [2^-127, 2^127]
            h_B_sc[i * NUM_BLOCKS + block] = __nv_fp8_e8m0(amax / DEST_MAX); 
            // printf("actual: %f, stored: %d, val: %f\n", amax / DEST_MAX, 
            //     *reinterpret_cast<uint8_t *>(&h_B_sc[i * NUM_BLOCKS + block]) - 127,
            //     powf(2., *reinterpret_cast<uint8_t *>(&h_B_sc[i * NUM_BLOCKS + block]) - 127));

            // Quantize
            for (int j = 0; j < Q_BLOCK; j++) {
                float quantized_fp32 = h_B[i * K + block * Q_BLOCK + j] / 
                    powf(2., *reinterpret_cast<uint8_t *>(&h_B_sc[i * NUM_BLOCKS + block]) - 127);
                h_B_fp8[i * K + block * Q_BLOCK + j] = __hip_fp8_e4m3_fnuz(quantized_fp32);
            }
        }
    }

    // Sanity check: dequantize and check errors
    // for (int i = 0; i < M; i++) {
    //     for (int block = 0; block < NUM_BLOCKS; block++) {
    //         for (int j = 0; j < Q_BLOCK; j++) {
    //             float dequantized_fp32 = float(h_A_fp8[i * K + block * Q_BLOCK + j]) * 
    //                 powf(2., *reinterpret_cast<uint8_t *>(&h_A_sc[i * NUM_BLOCKS + block]) - 127);
    //             float error = fabsf(h_A[i * K + block * Q_BLOCK + j] - dequantized_fp32);
    //             printf("A: %f, dequantized: %f, error: %f\n", h_A[i * K + block * Q_BLOCK + j], dequantized_fp32, error);
    //         }
    //     }
    // }
    // for (int i = 0; i < N; i++) {
    //     for (int block = 0; block < NUM_BLOCKS; block++) {
    //         for (int j = 0; j < Q_BLOCK; j++) {
    //             float dequantized_fp32 = float(h_B_fp8[i * K + block * Q_BLOCK + j]) * 
    //                 powf(2., *reinterpret_cast<uint8_t *>(&h_B_sc[i * NUM_BLOCKS + block]) - 127);
    //             float error = fabsf(h_B[i * K + block * Q_BLOCK + j] - dequantized_fp32);
    //             printf("B: %f, dequantized: %f, error: %f\n", h_B[i * K + block * Q_BLOCK + j], dequantized_fp32, error);
    //         }
    //     }
    // }

    // Run reference GEMM
    #pragma omp parallel for collapse(2)
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += h_A[i * K + k] * h_B[j * N + k];
            }
            h_C_ref[i * N + j] = sum;
        }
    }
    std::cout << "Performed CPU matrix multiplication" << std::endl;

    // Allocate device memory
    __hip_fp8_e4m3_fnuz *d_A_fp8;
    __hip_fp8_e4m3_fnuz *d_B_fp8;
    __nv_fp8_e8m0 *d_A_sc;
    __nv_fp8_e8m0 *d_B_sc;
    float *d_C;
    CUDACHECK(hipMalloc(&d_A_fp8, M * K * sizeof(__hip_fp8_e4m3_fnuz)));
    CUDACHECK(hipMalloc(&d_B_fp8, K * N * sizeof(__hip_fp8_e4m3_fnuz)));
    CUDACHECK(hipMalloc(&d_A_sc, M * K / Q_BLOCK * sizeof(__nv_fp8_e8m0)));
    CUDACHECK(hipMalloc(&d_B_sc, K / Q_BLOCK * N * sizeof(__nv_fp8_e8m0)));
    CUDACHECK(hipMalloc(&d_C, M * N * sizeof(float)));
    std::cout << "Allocated device memory" << std::endl;

    // Copy data to device
    CUDACHECK(hipMemcpy(d_A_fp8, h_A_fp8, M * K * sizeof(__hip_fp8_e4m3_fnuz), hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(d_B_fp8, h_B_fp8, N * K * sizeof(__hip_fp8_e4m3_fnuz), hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(d_A_sc, h_A_sc, M * NUM_BLOCKS * sizeof(__nv_fp8_e8m0), hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(d_B_sc, h_B_sc, N * NUM_BLOCKS * sizeof(__nv_fp8_e8m0), hipMemcpyHostToDevice));
    CUDACHECK(hipMemset(d_C, 999999999.0f, M * N * sizeof(float))); // useful for checking errors
    std::cout << "Copied data to device" << std::endl;

    // Create tensor map descriptor for matrix A
    constexpr int tma_dim = 5; // always use all 5 dimensions
    constexpr int swizzle_bytes = 32; // should change accordingly on TILE_K
    constexpr int swizzle_elements = swizzle_bytes / sizeof(__hip_fp8_e4m3_fnuz);
    constexpr CUtensorMapSwizzle tma_swizzle = 
        swizzle_bytes == 32  ? CU_TENSOR_MAP_SWIZZLE_32B  :
        swizzle_bytes == 64  ? CU_TENSOR_MAP_SWIZZLE_64B  :
        swizzle_bytes == 128 ? CU_TENSOR_MAP_SWIZZLE_128B : 
                               CU_TENSOR_MAP_SWIZZLE_NONE;
    static_assert(K % swizzle_elements == 0);
    CUtensorMap A_tmap;
    uint64_t A_gmem_shape [5] = {
        (uint64_t)swizzle_elements,
        (uint64_t)M, 
        (uint64_t)K / swizzle_elements, 
        1, 
        1
    };
    uint64_t A_gmem_stride[4] = {
        (uint64_t)K * sizeof(__hip_fp8_e4m3_fnuz), 
        (uint64_t)swizzle_bytes, 
        (uint64_t)M * K * sizeof(__hip_fp8_e4m3_fnuz), // never utilized
        (uint64_t)M * K * sizeof(__hip_fp8_e4m3_fnuz)  // never utilized
    };
    uint32_t A_smem_shape [5] = {
        swizzle_elements, 
        TILE_M, 
        TILE_K / swizzle_elements, 
        1, 
        1
    };
    uint32_t A_smem_stride[5] = {1, 1, 1, 1, 1};
    CUCHECK(cuTensorMapEncodeTiled(
        &A_tmap,
        CU_TENSOR_MAP_DATA_TYPE_UINT8, // there is no FP8 TMA type
        tma_dim,
        (void *)d_A_fp8,
        (uint64_t *)A_gmem_shape,
        (uint64_t *)A_gmem_stride, 
        (uint32_t *)A_smem_shape,
        (uint32_t *)A_smem_stride,
        CU_TENSOR_MAP_INTERLEAVE_NONE, // don't need this
        tma_swizzle,
        CU_TENSOR_MAP_L2_PROMOTION_NONE, // don't need this
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE // don't need this
    ));

    // Create tensor map descriptor for matrix B
    CUtensorMap B_tmap;
    uint64_t B_gmem_shape [5] = {
        (uint64_t)swizzle_elements,
        (uint64_t)N, 
        (uint64_t)K / swizzle_elements, 
        1, 
        1
    };
    uint64_t B_gmem_stride[4] = {
        (uint64_t)K * sizeof(__hip_fp8_e4m3_fnuz), 
        (uint64_t)swizzle_bytes, 
        (uint64_t)N * K * sizeof(__hip_fp8_e4m3_fnuz), // never utilized
        (uint64_t)N * K * sizeof(__hip_fp8_e4m3_fnuz)  // never utilized
    };
    uint32_t B_smem_shape [5] = {
        swizzle_elements, 
        TILE_N, 
        TILE_K / swizzle_elements, 
        1, 
        1
    };
    uint32_t B_smem_stride[5] = {1, 1, 1, 1, 1};
    CUCHECK(cuTensorMapEncodeTiled(
        &B_tmap,
        CU_TENSOR_MAP_DATA_TYPE_UINT8, // there is no FP8 TMA type
        tma_dim,
        (void *)d_B_fp8,
        (uint64_t *)B_gmem_shape,
        (uint64_t *)B_gmem_stride, 
        (uint32_t *)B_smem_shape,
        (uint32_t *)B_smem_stride,
        CU_TENSOR_MAP_INTERLEAVE_NONE, // don't need this
        tma_swizzle,
        CU_TENSOR_MAP_L2_PROMOTION_NONE, // don't need this
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE // don't need this
    ));

    // Launch kernel
    std::cout << "Launching kernel..." << std::endl;
    dim3 grid(SM_COUNT, 1, 1);
    dim3 block(NUM_THREADS, 1, 1);
    CUDACHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
        kernel), hipFuncAttributeMaxDynamicSharedMemorySize, DYNAMIC_SHARED_MEMORY
    ));

    // Warmup
    for (int i = 0; i < 5; i++) {
        kernel<<<grid, block, DYNAMIC_SHARED_MEMORY>>>(
            d_A_fp8, d_A_sc, A_tmap, d_B_fp8, d_B_sc, B_tmap, d_C);
        CUDACHECK(hipDeviceSynchronize());
    }

    // Benchmark
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 20; i++)
        kernel<<<grid, block, DYNAMIC_SHARED_MEMORY>>>(
            d_A_fp8, d_A_sc, A_tmap, d_B_fp8, d_B_sc, B_tmap, d_C);
    hipDeviceSynchronize(); // no CUDACHECK here
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::micro> diff = end - start;
    double duration_us = diff.count() / 20.0;
    std::cout << "Kernel execution time: " << duration_us << " us" << std::endl;

    // Calculate TFLOPs
    double flops = double(2.0) * M * N * K;
    double tflops = (flops / duration_us) / 1e6;
    std::cout << "Achieved performance: " << tflops << " TFLOPs" << std::endl;

    // Copy result back to host
    CUDACHECK(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "Copied results back to host" << std::endl;

    // Check results
    float max_error = 0.0f;
    float average_error = 0.0f;
    for (int i = 0; i < M * N; i++) {
        float error = std::abs(h_C[i] - h_C_ref[i]);
        max_error = std::max(max_error, error);
        average_error += error;
    }
    average_error /= M * N;
    std::cout << "Max error: " << max_error << std::endl;
    std::cout << "Average error: " << average_error << std::endl;

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_A_fp8;
    delete[] h_B_fp8;
    delete[] h_A_sc;
    delete[] h_B_sc;
    delete[] h_C;
    delete[] h_C_ref;
    hipFree(d_A_fp8);
    hipFree(d_B_fp8);
    hipFree(d_A_sc);
    hipFree(d_B_sc);
    hipFree(d_C);

    return 0;
}
