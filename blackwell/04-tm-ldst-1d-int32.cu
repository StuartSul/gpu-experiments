#include "hip/hip_runtime.h"
/*
    Tensor memory simple load/store

    Observations:
        - Each warp can only access 32 lanes of the TM
        - Thus, whatever TM address we pass, the lane value is ** AUTOMATICALLY mod-32'ed ** and added warp_id * 32.
        - For instance, if I pass in 0x0060'0000 to warp 0 (which refer to lane 96), it will just mod the lane and access lane 0
        - Likely, if I pass in lane 0 to warp 3, it will access lane 96
*/


#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;

// Kernel globals
struct globals {
    gl<int, 1, 1, 1, -1> tensor;

    __host__ inline dim3 grid() { return dim3(1); } // use single block
    __host__ inline dim3 block() { return dim3(128); } // use single warpgroup
    __host__ inline int dynamic_shared_memory() { return MAX_SHARED_MEMORY - 1024; }
};

// Kernel implementation
__global__ void kernel(const __grid_constant__ globals G) {
    // Allocate Tensor Memory (TM) for 1-CTA group 
    __shared__ uint32_t tm_addr_shared;
    uint32_t tm_addr = 0;
    uint32_t n_cols = 512; // full TM allocation
    if (threadIdx.x < 32) { // must be performed by a single warp in the CTA
        asm volatile("tcgen05.alloc.cta_group::1.sync.aligned.b32 [%0], %1;"
            :: "l"((uint64_t)&tm_addr_shared), "r"(n_cols)
        ); // __syncwarp() naturally happens here
        // After relinquish_alloc_permit, it becomes illegal for this CTA to call tcgen05.alloc
        asm volatile("tcgen05.relinquish_alloc_permit.cta_group::1.sync.aligned;");
    }
    __syncthreads();
    tm_addr = tm_addr_shared; // Move from shared memory into register

    // TM store launched by threads 0, ..., 127
    int src = threadIdx.x;
    asm volatile("tcgen05.st.sync.aligned.16x64b.x1.b32 [%0], {%1};"
        :: "r"(tm_addr + (3 - (threadIdx.x / 32)) * 0x0020'0000), "r"(src)
    );
    asm volatile("tcgen05.wait::st.sync.aligned;"); // waits for st issued by current thread
    asm volatile("bar.sync %0, %1;" :: "n"(0), "n"(128)); // warpgroup sync

    // TM load launched by threads 0, ..., 127
    int dst = 0; // to truly check if value got loaded
    asm volatile("tcgen05.ld.sync.aligned.16x64b.x1.b32 {%1}, [%0];"
        :: "r"(tm_addr + (threadIdx.x / 32) * 0x0020'0000), "r"(dst) // although different addr, mod 32 will make TM access per warp the same
    );
    asm volatile("tcgen05.wait::st.sync.aligned;"); // waits for st issued by current thread
    asm volatile("bar.sync %0, %1;" :: "n"(0), "n"(128)); // warpgroup sync

    // Save to global memory for validation
    G.tensor.raw_ptr[threadIdx.x] = dst;

    // De-allocate TM for 1-CTA group
    // Without this, CUDA will raise unfreed tensor memory error
    if (threadIdx.x < 32) { // must be performed by a single warp in the CTA
        asm volatile("tcgen05.dealloc.cta_group::1.sync.aligned.b32 %0, %1;"
            :: "r"(tm_addr), "r"(n_cols)
        );
    }
}

// Python bindings
PYBIND11_MODULE(_C, m) {
    kittens::py::bind_kernel<kernel>(m, "kernel",
        &globals::tensor
    );
}
