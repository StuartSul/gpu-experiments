#include "hip/hip_runtime.h"
/*
    Motivation:
        On H100s, it made sense to have multiple consumer warpgroups. Because the
        tensor core matrix multiply requires an entire warpgroup, we need to launch
        multiple matrix multiplies in asynchronously. 
        But on B200s, matrix multiplies only require a single thread. The 2 consumer
        warpgroups we used to have aren't really doing anything while the matmuls are
        happening. Also, the matmul epilogue operations are all synchronous (no chance
        for overlapping). So I wanted to know if it is beneficial to actually have 
        multiple consumer warpgroups in B200 matrix multiplies.

    Observations:
        - 
*/

#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;

static constexpr int _CONSUMER_WARPGROUPS = 1; // <-- changed this to 1 & 2

// Kernel configuration
struct config {
    static constexpr int SM_COUNT = 148;
    static constexpr int STATIC_SHARED_MEMORY = 1024;
    static constexpr int DYNAMIC_SHARED_MEMORY = MAX_SHARED_MEMORY - STATIC_SHARED_MEMORY;

    static constexpr int CONSUMER_WARPGROUPS = _CONSUMER_WARPGROUPS;
    static constexpr int PRODUCER_WARPGROUPS = 1;
    static constexpr int NUM_WARPGROUPS = CONSUMER_WARPGROUPS + PRODUCER_WARPGROUPS;
    static constexpr int NUM_WARPS = NUM_WARPGROUPS * WARPGROUP_WARPS;
    static constexpr int NUM_THREADS = NUM_WARPS * WARP_THREADS;

    static constexpr int PRODUCER_REGISTERS = 40;
    static constexpr int CONSUMER_REGISTERS = 232;

    static constexpr int PIPELINE_STAGES = 4;
};

// Kernel globals
struct globals {
    static constexpr int SUPERGROUP_BLOCKS = 12;
    static constexpr int ROW_BLOCK = 128;
    static constexpr int COL_BLOCK = 128;
    static constexpr int REDUCTION_BLOCK = 128;

    using A_tile = st_fp8e4m3<ROW_BLOCK, REDUCTION_BLOCK>;
    using B_tile = st_fp8e4m3<COL_BLOCK, REDUCTION_BLOCK>;
    using C_tile = st_bf<ROW_BLOCK, COL_BLOCK>;

    gl<fp8e4m3, 1, 1, -1, -1, A_tile> A;
    gl<fp8e4m3, 1, 1, -1, -1, B_tile> B;
    gl<bf16, 1, 1, -1, -1, C_tile> C;

    __host__ inline dim3 grid() { return dim3(config::SM_COUNT); }
    __host__ inline dim3 block() { return dim3(config::NUM_THREADS); }
    __host__ inline int dynamic_shared_memory() { return config::DYNAMIC_SHARED_MEMORY; }

    struct pipeline_inputs {
        A_tile A;
        B_tile B;
    };

    struct pipeline_outputs {
        C_tile C;
    };
};

// Kernel implementation
__global__ __launch_bounds__(config::NUM_THREADS, 1)
void kernel(const __grid_constant__ globals G) {
    // Shared memory declaration
    extern __shared__ int __shm[];
    tma_swizzle_allocator allocator((int*)&__shm[0]);

    // Warpgroup configuration
    using consumer = group<config::CONSUMER_WARPGROUPS * WARPGROUP_WARPS>;
    int warpgroup_id = warpgroup::groupid();
    int warp_id = warpgroup::warpid();
    int lane_id = warp::laneid();

    // Allocate shared and tensor memory
    static_assert(sizeof(globals::pipeline_inputs) * config::PIPELINE_STAGES + sizeof(globals::pipeline_outputs) <= config::DYNAMIC_SHARED_MEMORY);
    globals::pipeline_inputs (&inputs)[config::PIPELINE_STAGES] = allocator.allocate<globals::pipeline_inputs, config::PIPELINE_STAGES>();
    globals::pipeline_outputs &outputs = allocator.allocate<globals::pipeline_outputs>();
    tensor_allocator<1, 1> tm_allocator {};

    // Set up mbarriers
    __shared__ semaphore inputs_arrived[config::PIPELINE_STAGES];
    __shared__ semaphore inputs_finished[config::PIPELINE_STAGES];
    __shared__ semaphore outputs_arrived;
    __shared__ semaphore tensors_finished;
    if (threadIdx.x == 0) {
        for (int i = 0; i < config::PIPELINE_STAGES; ++i) {
            init_semaphore(inputs_arrived[i], 0, 1);
            init_semaphore(inputs_finished[i], 0, 1);
        }
        init_semaphore(outputs_arrived, 0, 1);
        init_semaphore(tensors_finished, 0, 1);
    }
    __syncthreads();

    // Pipeline configuration
    int num_blocks_per_row = G.C.cols() / globals::COL_BLOCK;
    int num_blocks_per_col = G.C.rows() / globals::ROW_BLOCK;
    int num_blocks = num_blocks_per_row * num_blocks_per_col;
    int num_iters_per_block = G.A.cols() / globals::REDUCTION_BLOCK;
    int num_blocks_per_supergroup = globals::SUPERGROUP_BLOCKS * num_blocks_per_row;

    // Declare stage and phasebits for semaphore waits
    int stage = 0;
    int last_stage = -1;
    uint32_t phasebits = 0xFFFF0000;

    // Main divergence
    if (warpgroup_id == config::NUM_WARPGROUPS - 1) {
        // Producer group
        warpgroup::decrease_registers<config::PRODUCER_REGISTERS>();

        // Sub divergence
        if (warp_id == 3 && lane_id == 0) {
            // Producer group -- loaders
            for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
                // Compute block indices
                int supergroup_idx = block_idx / num_blocks_per_supergroup;
                int idx_within_supergroup = block_idx % num_blocks_per_supergroup;
                int rows_in_supergroup = min(globals::SUPERGROUP_BLOCKS, num_blocks_per_col - supergroup_idx * globals::SUPERGROUP_BLOCKS);
                int row_within_supergroup = idx_within_supergroup % rows_in_supergroup;
                int row_block_idx = supergroup_idx * globals::SUPERGROUP_BLOCKS + row_within_supergroup;
                int col_block_idx = idx_within_supergroup / rows_in_supergroup;

                for (int i = 0; i < num_iters_per_block; ++i) {
                    wait(inputs_finished[stage], get_phasebit<1>(phasebits, stage));
                    if (stage == last_stage) {
                        arrive(outputs_arrived);
                        last_stage = -1;
                    }
                    tma::expect_bytes(inputs_arrived[stage], sizeof(globals::pipeline_inputs));
                    tma::load_async(inputs[stage].A, G.A, {row_block_idx, i}, inputs_arrived[stage]);
                    tma::load_async(inputs[stage].B, G.B, {col_block_idx, i}, inputs_arrived[stage]);
                    update_phasebit<1>(phasebits, stage);
                    if (i == num_iters_per_block - 1) {
                        last_stage = stage;
                    }
                    stage = (stage + 1) % config::PIPELINE_STAGES;
                }
            }
            wait(inputs_finished[last_stage], get_phasebit<1>(phasebits, last_stage));
            arrive(outputs_arrived);
        } else if (warp_id == 0 && lane_id == 0) {
            // Producer group -- launchers
            using tm_t = tt<float, globals::ROW_BLOCK, globals::COL_BLOCK>;
            tm_t tm = tm_allocator.template allocate<tm_t>(0);
            for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
                wait(tensors_finished, get_phasebit<1>(phasebits, config::PIPELINE_STAGES));
                update_phasebit<1>(phasebits, config::PIPELINE_STAGES);
                wait(inputs_arrived[stage], get_phasebit<0>(phasebits, stage));
                update_phasebit<0>(phasebits, stage);
                mm_ABt(tm, inputs[stage].A, inputs[stage].B, inputs_finished[stage]);
                stage = (stage + 1) % config::PIPELINE_STAGES;
                for (int i = 1; i < num_iters_per_block; ++i) {
                    wait(inputs_arrived[stage], get_phasebit<0>(phasebits, stage)); 
                    update_phasebit<0>(phasebits, stage);
                    mma_ABt(tm, inputs[stage].A, inputs[stage].B, inputs_finished[stage]);
                    stage = (stage + 1) % config::PIPELINE_STAGES;
                }
            }
        }
    } else if (warpgroup_id < config::CONSUMER_WARPGROUPS) {
        // Consumer group
        warpgroup::increase_registers<config::CONSUMER_REGISTERS>();
        using tm_t = tt<float, globals::ROW_BLOCK, globals::COL_BLOCK>;
        tm_t tm = tm_allocator.template allocate<tm_t>(0);

        for (int block_idx = blockIdx.x; block_idx < num_blocks; block_idx += gridDim.x) {
            // Compute block indices
            int supergroup_idx = block_idx / num_blocks_per_supergroup;
            int idx_within_supergroup = block_idx % num_blocks_per_supergroup;
            int rows_in_supergroup = min(globals::SUPERGROUP_BLOCKS, num_blocks_per_col - supergroup_idx * globals::SUPERGROUP_BLOCKS);
            int row_within_supergroup = idx_within_supergroup % rows_in_supergroup;
            int row_block_idx = supergroup_idx * globals::SUPERGROUP_BLOCKS + row_within_supergroup;
            int col_block_idx = idx_within_supergroup / rows_in_supergroup;

            // Wait for the last matmul to complete
            wait(outputs_arrived, get_phasebit<0>(phasebits, config::PIPELINE_STAGES));
            update_phasebit<0>(phasebits, config::PIPELINE_STAGES);

            // Load the output from tensor memory into registers
            rt_fl<globals::ROW_BLOCK / (4 * config::CONSUMER_WARPGROUPS), globals::COL_BLOCK> C_reg;
            consumer::load_async(C_reg, tm);
            tensor_load_wait();
            consumer::sync(0);
            if (consumer::laneid() == 0)
                arrive(tensors_finished);

            // Store to global memory
            consumer::store(outputs.C, C_reg);
            consumer::sync(0);
            consumer::tma::store_async(G.C, outputs.C, {row_block_idx, col_block_idx});
            tma::store_async_read_wait();
            consumer::sync(0);
        }
    }
}

// Python bindings
PYBIND11_MODULE(_C, m) {
    m.doc() = "";
    kittens::py::bind_kernel<kernel>(m, "kernel",
        &globals::A,
        &globals::B,
        &globals::C
    );
}
