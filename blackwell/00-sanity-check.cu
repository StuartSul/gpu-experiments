#include "hip/hip_runtime.h"
#include "gpu-experiments.cuh"

__global__ void kernel() {
    printf("Block: %d/%d | Thread: %d/%d\n", blockIdx.x, gridDim.x, threadIdx.x, blockDim.x);
}

int main() {
    kernel<<<148, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
