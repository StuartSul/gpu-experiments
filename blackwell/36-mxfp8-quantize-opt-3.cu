#include "hip/hip_runtime.h"
/*
    Observation 1:
        DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1) 
        DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1)
        DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1) 
        DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1) DO NOT USE CLUSTER_DIM(1)

        It was the reason why the kernels suddenly became slow! Like REALLY slow, like 40% slower!
        It changes the way CUDA schedules blocks in a way that reduces the occupancy. Probably because
        it schedules at the granularity of clusters instead of individual blocks. I think there is also
        extra launch setup/teardown overhead for cluster mode.

    Observation 2:
        cp.async.bulk is surprisingly slow compared to cp.async.bulk.tensor, even if we are storing 1D tensor.
        Thus, we should just always use cp.async.bulk.tensor, and never cp.async.bulk

    Observation 3:
        - Let's fix to 204800x2048 matrix, 128x128 tile for this
        - Without calculation logic, we reach 6075.39 GB/s
        - The full thing reaches THE SAME BANDWIDTH. Now we are truly memory bound!!
            - Remember that a little slowdown happens inevitably since calculation logic
              consumes a bit of shared memory bandwidth
    
    Observation 4 (hypothesis, not fully verified):
        - The TK style of "load everything from smem" -> "do it" -> "store everything to smem"
          only works when kernel is compute bound.
        - With memory-bound kernels, this is no longer the case. We should do little bit of 
          shared memory load, calculation, then little bit of shared memory store.
        - Why? Because we don't want to clog up shared memory bandwidth in memory bound workloads.
          We need to ensure gmem <-> smem is always fully saturated, and smem <-> reg should 
          bother this as little as possible.
        ** Note: I changed the load part to full load, and speed wasn't changed. Maybe this isn't the case **
    
    Observation 5
        - Bank conflict is a real thing
        - Changing just the load pattern to non-swizzled degrades perf down to 4030 GB/s
        - Changing load and store pattern to non-swizzled (thus result correct) degrades perf down to 2901 GB/s. 2x slower!
*/

#include <kittens.cuh>
#include <pybind11/pybind11.h>

using namespace kittens;
namespace py = pybind11;

static constexpr int M = 204800;
static constexpr int N = 2048;

// Changing these requires re-writing the kernel
static constexpr int TILE_M = 128;
static constexpr int TILE_N = 128;
static constexpr int Q_BLOCK_SIZE = 32;

__global__ __launch_bounds__(TILE_M)
void kernel(
    const bf16 *A_bf16,
    const __grid_constant__ CUtensorMap A_bf16_tmap,
    fp8e4m3 *A_fp8,
    const __grid_constant__ CUtensorMap A_fp8_tmap,
    fp8e8m0 *A_sc,
    const __grid_constant__ CUtensorMap A_sc_tmap
) {
    // Allocate shared memory
    extern __shared__ int __shm[];
    uint64_t __shm_base = reinterpret_cast<uint64_t>(&__shm[0]);
    bf16 *A_bf16_smem = reinterpret_cast<bf16*>(((__shm_base + 1023) / 1024) * 1024);
    fp8e4m3 *A_fp8_smem = reinterpret_cast<fp8e4m3*>(A_bf16_smem);
    fp8e8m0 *A_sc_smem = reinterpret_cast<fp8e8m0*>(A_fp8_smem + TILE_M * TILE_N); // naturally fulfills alignment

    // Initialize mbarriers
    __shared__ semaphore inputs_arrived;
    if (threadIdx.x == 0) {
        init_semaphore(inputs_arrived, 0, 1);
    }
    __syncthreads();

    // Calculate tile index
    int row = blockIdx.y * TILE_M;
    int col = blockIdx.x * TILE_N;

    // Initiate the load from global memory
    if (threadIdx.x == 0) {
        tma::expect_bytes(inputs_arrived, TILE_M * TILE_N * sizeof(bf16));
        asm volatile("{cp.async.bulk.tensor.2d.shared::cta.global.tile.mbarrier::complete_tx::bytes.cta_group::1 [%0], [%1, {%2, %3}], [%4];}"
            :: "l"(__cvta_generic_to_shared(A_bf16_smem)), "l"(&A_bf16_tmap), "r"(col), "r"(row), "l"(__cvta_generic_to_shared(&inputs_arrived))
            : "memory");
    }

    // Wait for the load to complete
    asm volatile("{fence.proxy.async.shared::cta;}" ::: "memory"); // make writes to smem visible
    wait(inputs_arrived, 0);

    // We have 128 threads per block. Each thread handles a row of 128 elements
    constexpr int NUM_Q_BLOCKS = TILE_N / Q_BLOCK_SIZE; // 4
    constexpr int N_PER_Q_BLOCK = TILE_N / 2 / NUM_Q_BLOCKS; // 16
    bf16_2 A_bf16_reg[NUM_Q_BLOCKS][N_PER_Q_BLOCK];
    fp8e8m0 A_sc_reg[NUM_Q_BLOCKS];

    // Load input matrix from shared memory (swizzled)
    #pragma unroll
    for (int i = 0; i < NUM_Q_BLOCKS; i++) {
        int q_block_idx = (i + threadIdx.x / 8) % NUM_Q_BLOCKS;
        #pragma unroll
        for (int j = 0; j < N_PER_Q_BLOCK; j++) {
            int offset = threadIdx.x * TILE_N * sizeof(bf16) + // row
                         q_block_idx * Q_BLOCK_SIZE  * sizeof(bf16) + // Q block
                         ((threadIdx.x + j) % 16) * sizeof(bf16_2); // element within Q block (swizzled)
            asm volatile("{ld.shared.b32 %0, [%1];}"
                : "=r"(*reinterpret_cast<uint32_t *>(&A_bf16_reg[i][j]))
                : "r"(static_cast<uint32_t>(__cvta_generic_to_shared(A_bf16_smem)) + offset));
        }
    }
    __syncthreads();

    // Perform MXFP8 quantization
    #pragma unroll
    for (int i = 0; i < NUM_Q_BLOCKS; i++) {
        // A group of 8 threads handles the same Q block segment
        int q_block_idx = (i + threadIdx.x / 8) % NUM_Q_BLOCKS;

        // Calculate absolute maximum
        bf16_2 amax = __habs2(A_bf16_reg[i][0]);
        #pragma unroll
        for (int j = 1; j < N_PER_Q_BLOCK; j++)
            amax = __hmax2(amax, __habs2(A_bf16_reg[i][j]));

        // Compute the scales
        // Must narrow to e8m0, rounding towards positive infinity and saturating to finite, then clamp
        // https://arxiv.org/pdf/2506.08027
        float scale = max(__bfloat162float(__hmax(amax.x, amax.y)) * 0.002232142857f, 0.000000000001f);
        A_sc_reg[q_block_idx].__x = __nv_cvt_float_to_e8m0(scale, __HIP_SATFINITE, hipRoundPosInf); // causes stack frame, but ignorable
        scale = static_cast<float>(A_sc_reg[q_block_idx]); // utilizes the float() operator defined in __nv_fp8x2_e8m0

        // Quantize input matrix and store to share memory
        #pragma unroll
        for (int j = 0; j < N_PER_Q_BLOCK; j++) {
            int offset = threadIdx.x * TILE_N * sizeof(fp8e4m3) + // row
                         q_block_idx * Q_BLOCK_SIZE * sizeof(fp8e4m3) + // Q block
                         ((threadIdx.x + j) % 16) * sizeof(fp8e4m3_2); // element within Q block (swizzled)
            fp8e4m3 A_fp8_reg[2] = {
                __hip_fp8_e4m3_fnuz(__bfloat162float(A_bf16_reg[i][j].x) / scale),
                __hip_fp8_e4m3_fnuz(__bfloat162float(A_bf16_reg[i][j].y) / scale)
            };
            asm volatile("{st.shared.b16 [%0], %1;}"
                :: "r"(static_cast<uint32_t>(__cvta_generic_to_shared(A_fp8_smem)) + offset)
                   "h"(*reinterpret_cast<uint16_t *>(&A_fp8_reg[0])));
        }
    }

    // Store the scales to shared memory. Each thread will access 1 bank, so no need to swizzle,
    // but we do have to follow this complicated layout pattern made by NVIDIA:
    // https://docs.nvidia.com/cuda/parallel-thread-execution/#tcgen05-mma-scale-factor-a-layout-1x
    int scale_offset = (threadIdx.x % 32) * 16 + // row
                       (threadIdx.x / 32) * 4; // column
    asm volatile("{st.shared.b32 [%0], %1;}" 
        :: "r"(static_cast<uint32_t>(__cvta_generic_to_shared(A_sc_smem)) + scale_offset)
           "r"(*reinterpret_cast<uint32_t *>(&A_sc_reg[0])));

    // Store to global memory
    asm volatile("{fence.proxy.async.shared::cta;}" ::: "memory"); // make writes to smem visible
    __syncthreads();
    if (threadIdx.x == 0) {
        // Since this is the only store, no need to wait for completion
        asm volatile("{cp.async.bulk.tensor.2d.global.shared::cta.tile.bulk_group [%0, {%1, %2}], [%3];}"
            :: "l"(&A_fp8_tmap), "r"(col), "r"(row), 
               "r"(static_cast<uint32_t>(__cvta_generic_to_shared(A_fp8_smem)))
            : "memory");
        asm volatile("{cp.async.bulk.tensor.3d.global.shared::cta.tile.bulk_group [%0, {%1, %2, %3}], [%4];}"
            :: "l"(&A_sc_tmap), "n"(0), "r"(col / TILE_N), "r"(row / TILE_M), 
               "r"(static_cast<uint32_t>(__cvta_generic_to_shared(A_sc_smem)))
            : "memory");
        asm volatile("{cp.async.bulk.tensor.3d.global.shared::cta.tile.bulk_group [%0, {%1, %2, %3}], [%4];}"
            :: "l"(&A_sc_tmap), "r"(TILE_N * TILE_M / Q_BLOCK_SIZE / 2), "r"(col / TILE_N), "r"(row / TILE_M), 
               "r"(static_cast<uint32_t>(__cvta_generic_to_shared(A_sc_smem)) + TILE_M * TILE_N / Q_BLOCK_SIZE / 2)
            : "memory");
    }
}

template <typename T>
__host__ static inline T *get_data_ptr(py::object tensor) {
    // Assumes the following about `tensor`
    // - is a torch.Tensor object
    // - is contiguous
    // - is on device
    // - has the correct shape
    return reinterpret_cast<T *>(tensor.attr("data_ptr")().cast<uintptr_t>());
}

__host__ static inline void launch_kernel(py::object &A_bf16, py::object &A_fp8, py::object &A_sc) {
    CUtensorMap A_bf16_tmap, A_fp8_tmap, A_sc_tmap;

    static constexpr int A_bf16_rank = 2;
    static constexpr int A_fp8_rank = 2;
    static constexpr int A_sc_rank = 3;

    uint64_t A_bf16_shape[A_bf16_rank] = {N, M}; // inner-dim first
    uint64_t A_bf16_stride[A_bf16_rank - 1] = {N * sizeof(bf16)};
    uint32_t A_bf16_smem_shape[A_bf16_rank] = {TILE_N, TILE_M};
    uint32_t A_bf16_smem_stride[A_bf16_rank] = {1, 1};
    
    uint64_t A_fp8_shape[A_fp8_rank] = {N, M};
    uint64_t A_fp8_stride[A_fp8_rank - 1] = {N * sizeof(fp8e4m3)};
    uint32_t A_fp8_smem_shape[A_fp8_rank] = {TILE_N, TILE_M};
    uint32_t A_fp8_smem_stride[A_fp8_rank] = {1, 1};

    uint64_t A_sc_shape[A_sc_rank] = {TILE_N * TILE_M / Q_BLOCK_SIZE, N / TILE_N, M / TILE_M};
    uint64_t A_sc_stride[A_sc_rank - 1] = {TILE_N * TILE_M / Q_BLOCK_SIZE * sizeof(fp8e8m0), N * TILE_M / Q_BLOCK_SIZE * sizeof(fp8e4m3)};
    uint32_t A_sc_smem_shape[A_sc_rank] = {TILE_N * TILE_M / Q_BLOCK_SIZE / 2, 1, 1}; // divide into 2 TMA stores
    uint32_t A_sc_smem_stride[A_sc_rank] = {1, 1, 1};

    CUCHECK(cuTensorMapEncodeTiled(
        &A_bf16_tmap,
        CU_TENSOR_MAP_DATA_TYPE_BFLOAT16,
        A_bf16_rank,
        (void *)get_data_ptr<bf16>(A_bf16),
        &A_bf16_shape[0],
        &A_bf16_stride[0],
        &A_bf16_smem_shape[0],
        &A_bf16_smem_stride[0],
        CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_NONE,
        CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    ));
    CUCHECK(cuTensorMapEncodeTiled(
        &A_fp8_tmap,
        CU_TENSOR_MAP_DATA_TYPE_UINT8,
        A_fp8_rank,
        (void *)get_data_ptr<fp8e4m3>(A_fp8),
        &A_fp8_shape[0],
        &A_fp8_stride[0],
        &A_fp8_smem_shape[0],
        &A_fp8_smem_stride[0],
        CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_NONE,
        CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    ));
    CUCHECK(cuTensorMapEncodeTiled(
        &A_sc_tmap,
        CU_TENSOR_MAP_DATA_TYPE_UINT8,
        A_sc_rank,
        (void *)get_data_ptr<fp8e8m0>(A_sc),
        &A_sc_shape[0],
        &A_sc_stride[0],
        &A_sc_smem_shape[0],
        &A_sc_smem_stride[0],
        CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_NONE,
        CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    ));

    static constexpr int DYNAMIC_SMEM = TILE_M * TILE_N * sizeof(bf16) + 1024;
    dim3 grid = dim3(N / TILE_N, M / TILE_M);
    CUDACHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, DYNAMIC_SMEM));
    kernel<<<grid, TILE_M, DYNAMIC_SMEM>>>(
        get_data_ptr<bf16>(A_bf16), A_bf16_tmap, get_data_ptr<fp8e4m3>(A_fp8), A_fp8_tmap, get_data_ptr<fp8e8m0>(A_sc), A_sc_tmap
    );
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("kernel", &launch_kernel);
}
