#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void atomicAndVolatileKernel(int* sign, int* data) {
    if (*(volatile int*)sign == 0)
        atomicAdd(data, 1);
}

int main() {
    int *d_sign, *d_data;
    
    hipMalloc(&d_sign, sizeof(int));
    hipMalloc(&d_data, sizeof(int));
    
    hipMemset(d_sign, 0, sizeof(int));
    hipMemset(d_data, 0, sizeof(int));
    
    atomicAndVolatileKernel<<<1, 1>>>(d_sign, d_data);
    hipDeviceSynchronize();

    int h_data;
    hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost);
    printf("Data value: %d\n", h_data);

    hipFree(d_sign);
    hipFree(d_data);
    
    return 0;
}
